// Libraries
#include <fstream>
#include <iostream>
#include <string>
#include <limits>
#include <iomanip>

// THRUST
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// Header files
#include "io.cuh"
#include "../helper/Timer.cuh"

// Namespace
using namespace staccato;

// Reads dense Mtx file
void io::readMtxDense(thrust::host_vector<hipDoubleComplex> &A, std::string _filepath, std::string _filename, bool _isComplex){
    // Local variables
    size_t rowSize, colSize, entrySize;
    double _real, _imag;
    // Open file
    std::ifstream input;
    input.open(_filepath + _filename);
    //input.precision(std::numeric_limits<float>::digits8);
    // Ignore first line
    while (input.peek() == '%') input.ignore(2048, '\n');
    // Get matrix dimension
    input >> rowSize >> colSize;
    entrySize = rowSize * colSize;

    if (!input){
        std::cout << "File not found." << std::endl;
        exit(1);
    }
    else {
        //std::cout << ">> Reading matrix from "<< _filepath + _filename << " ... " << std::endl;
        //std::cout << ">> Matrix size: " << rowSize << " x " << colSize << std::endl;
        A.resize(entrySize+1);	// Causes segmentation fault without +1
        timerIO.start();
        // Complex matrix
        if (_isComplex){
            //std::cout << ">> Matrix type: COMPLEX" << std::endl;
            int i = 0;
            while (!input.eof()) {
                input >> _real >> _imag;
                hipDoubleComplex temp;
                temp.x = _real;
                temp.y = _imag;
                A[i] = temp;
                i++;
            }
            timerIO.stop();
            //std::cout << ">> Matrix " << _filename << " read" << std::endl;
            //std::cout << ">>>> Time taken = " << timerIO.getDurationMicroSec()*1e-6 << "\n" << std::endl;
        }
        // Real matrix
        else if (!_isComplex){
            //std::cout << ">> Matrix type: REAL" << std::endl;
            int i = 0;
            while (!input.eof()) {
                input >> _real >> _imag;
                hipDoubleComplex temp;
                temp.x = _real;
                temp.y = _imag;
                A[i] = temp;
                i++;
            }
            timerIO.stop();
            //std::cout << ">> Matrix " << _filename << " read" << std::endl;
            //std::cout << ">>>> Time taken = " << timerIO.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;
        }
    }
    input.close();
}

// Writes solution vector
void io::writeSolVecComplex(thrust::host_vector<hipDoubleComplex> &sol, std::string _filepath, std::string _filename){
    std::ofstream output;
    output.open(_filepath + _filename);
    timerIO.start();
    // Write header
    if (!output.is_open()){
        std::cout << ">> ERROR: Unable to open output file for solution vector" << std::endl;
    }
    else{
        output << std::setw(25) << std::left << "Real" << "    ";
        output << std::setw(25) << std::left << "Imag" << "\r\n";
        // Write data
        for (size_t i = 0; i < sol.size(); i++){
            output << std::setprecision(16) << std::setw(25) << std::left << sol[i].x << "    ";
            output << std::setprecision(16) << std::setw(25) << std::left << sol[i].y << "\r\n";
        }
    }
    // Close file
    timerIO.stop();
    // Output messages
    std::cout << ">> Solution vector written in " << _filepath + _filename << std::endl;
    std::cout << ">>>> Vector size = " << sol.size() << std::endl;
    std::cout << ">>>> Time taken = " << timerIO.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;
}
