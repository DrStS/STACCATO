// Libraries
#include <iostream>
#include <string>
#include <cmath>

// OpenMP
#include <omp.h>

// THRUST
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// CUBLAS
#include <hipblas.h>

// NVTX
#include <nvToolsExt.h>

// Header files
#include "config/config.cuh"
#include "io/io.cuh"
#include "data/dataStructures.cuh"
#include "solver/assembly.cuh"
#include "helper/Timer.cuh"
#include "helper/helper.cuh"

// Definitions
#define	PI	3.14159265359
#define MAX_NUM_THREADS 32

// Pinned Allocators
typedef thrust::system::cuda::experimental::pinned_allocator<int> pinnedAllocInt;
typedef thrust::system::cuda::experimental::pinned_allocator<hipDoubleComplex> pinnedAlloc;
typedef thrust::system::cuda::experimental::pinned_allocator<hipDoubleComplex*> pinnedAllocPtr;

// Namespace
using namespace staccato;

/*
    DOFs

    i8: 2369456
    test set: 10914300
*/

int main (int argc, char *argv[]){

    timerTotal.start();

    PUSH_RANGE("Initial Configuration (Host)", 1)
    timerInit.start();

    /*--------------------
    COMMAND LINE ARGUMENTS
    --------------------*/
    double freq_max;
    int mat_repetition, subComponents, num_streams, num_threads, batchSize;
    // Configure test environment with command line arguments
    config::configureTest(argc, argv, freq_max, mat_repetition, subComponents, num_streams, num_threads, batchSize);

    /*---------------------
    FILEPATHS AND FILENAMES
    ---------------------*/
    // Vector of filepaths
    std::string filepath[2];
    filepath[0] = "/opt/software/examples/MOR/r_approx_180/\0";
    filepath[1] = "/opt/software/examples/MOR/r_approx_300/\0";
    // Solution filepath
    std::string filepath_sol = "output/";
    // Solution filename
    std::string filename_sol = "solution.dat";
    // Array of filenames
    std::string baseName_K = "KSM_Stiffness_r\0";
    std::string baseName_M = "KSM_Mass_r\0";
    std::string baseName_D = "KSM_Damping_r\0";
    std::string base_format = ".mtx\0";
    std::string filename_K[12], filename_M[12], filename_D[12];

    /*--------
    PARAMETERS
    --------*/
    double alpha = 4*PI*PI;
    hipDoubleComplex rhs_val;
    rhs_val.x = 1.0;
    rhs_val.y = 0.0;
    // Array of matrix sizes (row)
    int row_baseline[] = {126, 132, 168, 174, 180, 186, 192, 288, 294, 300, 306, 312};
    // Frequency vector
    thrust::host_vector<int, pinnedAllocInt> freq(batchSize), freq_square(batchSize);
    // Fill in frequency vectors
    thrust::sequence(freq.begin(), freq.end(), 1);
    thrust::transform(freq.begin(), freq.end(), freq.begin(), freq_square.begin(), thrust::multiplies<int>());

    /*----------------------------
    OPENMP & CUBLAS INITIALIZATION
    ----------------------------*/
    // OpenMP
    int tid;
    omp_set_num_threads(num_threads);
    omp_set_dynamic(0);
    omp_set_nested(1);
    omp_set_num_threads(num_threads);
    // cuBLAS
    hipblasHandle_t cublasHandle[MAX_NUM_THREADS];
    for (size_t i = 0; i < num_threads; ++i) hipblasCreate(cublasHandle + i);

    /*-----------------------
    CHECK MEMORY REQUIREMENTS
    -----------------------*/
    config::check_memory(mat_repetition, freq_max, num_threads);

    timerInit.stop();
    std::cout << ">> Initial Configuration done" << std::endl;
    std::cout << ">>>> Time taken = " << timerInit.getDurationMicroSec()*1e-6 << " sec" << "\n" << std::endl;
    POP_RANGE; // Initial Configuration

    /*--------------------
    DATA STRUCTURES (HOST)
    --------------------*/
    PUSH_RANGE("Data Structures (Host)", 1)
    timerDataHost.start();
    // Create matrix host_vectors
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub(12), M_sub(12), D_sub(12);
    thrust::host_vector<hipDoubleComplex> K, M, D;

    // Array information
    thrust::host_vector<int> row_sub(subComponents), nnz_sub(subComponents);
    thrust::host_vector<int> shift_local_A(subComponents), shift_local_rhs(subComponents);
    int nnz, row, nnz_max;
    // Set up host data structures
    data::constructHostDataStructure(filename_K, filename_M, filename_D, filepath,
                                     baseName_K, baseName_M, baseName_D, base_format, row_baseline,
                                     K_sub, M_sub, D_sub);
    data::getInfoHostDataStructure(shift_local_A, shift_local_rhs,
                                   row_sub, nnz_sub, nnz,
                                   row, nnz_max,
                                   mat_repetition, row_baseline);
    data::combineHostMatrices(K_sub, M_sub, D_sub, K, M, D, nnz, mat_repetition, nnz_sub);

    timerDataHost.stop();
    std::cout << ">> Host data structure constructed" << std::endl;
    std::cout << ">>>> Time taken = " << timerDataHost.getDurationMicroSec()*1e-6 << " sec" << "\n" << std::endl;
    POP_RANGE // Data Structures (Host)

    /*----------------------
    DATA STRUCTURES (DEVICE)
    ----------------------*/
    PUSH_RANGE("Data Structures (Device)", 2)
    timerDataDevice.start();
    // Send matrices to device
    thrust::device_vector<hipDoubleComplex> d_K = K;
    thrust::device_vector<hipDoubleComplex> d_M = M;
    thrust::device_vector<hipDoubleComplex> d_D = D;
    // Create RHS vector directly on device (will be replaced with send operation)
    thrust::device_vector<hipDoubleComplex> d_rhs(row*freq_max, rhs_val);
    // Create matrix device_vectors
    thrust::device_vector<hipDoubleComplex> d_A_batch(num_threads*freq_max*nnz_max);
    // Get raw pointers to device matrices & vectors
    hipDoubleComplex *d_ptr_K_base       = thrust::raw_pointer_cast(d_K.data());
    hipDoubleComplex *d_ptr_M_base       = thrust::raw_pointer_cast(d_M.data());
    hipDoubleComplex *d_ptr_D_base       = thrust::raw_pointer_cast(d_D.data());
    hipDoubleComplex *d_ptr_A_batch_base = thrust::raw_pointer_cast(d_A_batch.data());
    hipDoubleComplex *d_ptr_rhs_base     = thrust::raw_pointer_cast(d_rhs.data());
    // Create device vectors of pointers for each sub-components from combined matrices on device
    thrust::device_vector<hipDoubleComplex*> d_ptr_K(subComponents), d_ptr_M(subComponents), d_ptr_D(subComponents);
    // Get information from device data structures
    data::getInfoDeviceDataStructure(d_ptr_K, d_ptr_M, d_ptr_D, d_ptr_K_base, d_ptr_M_base, d_ptr_D_base, nnz_sub, subComponents);

    timerDataDevice.stop();
    std::cout << ">> Device data structure constructed" << std::endl;
    std::cout << ">>>> Time taken = " << timerDataDevice.getDurationMicroSec()*1e-6 << " sec" << "\n" << std::endl;
    POP_RANGE // Data Structures (Device)

    /*--------------------------------
    Krylov Subspace Method Preparation
    --------------------------------*/
    PUSH_RANGE("Krylov Subspace Method Preparation", 2)
    timerMORprep.start();
    // M = 4*pi^2*M
    cublas_check(hipblasZdscal(cublasHandle[0], nnz, &alpha, d_ptr_M_base, 1));
    // Solver Info for batched LU decomposition
    thrust::device_vector<int> d_solverInfo(batchSize);
    int *d_ptr_solverInfo = thrust::raw_pointer_cast(d_solverInfo.data());
    int solverInfo_solve;
    // Stream initialisation
    hipStream_t streams[num_streams];
    for (size_t i = 0; i < num_streams; ++i){
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
        std::cout << ">> Stream " << i << " created" << std::endl;
    }
    timerMORprep.stop();
    std::cout << "\n>> Ready to start to Krylov Subspace Method" << std::endl;
    std::cout << ">>>> Time taken = " << timerMORprep.getDurationMicroSec()*1e-6 << " sec" << "\n" << std::endl;
    POP_RANGE // Krylov Subspace Method Preparation

    /*--------------------
    Krylov Subspace Method
    --------------------*/
    PUSH_RANGE("Krylov Subspace Method", 3)
    timerMOR.start();
    std::cout << ">> Krylov Subspace Method started" << std::endl;
#pragma omp parallel private(tid) num_threads(num_threads)
    {
        // Get thread number
        tid = omp_get_thread_num();
        // Allocate vector of array pointers to A in each thread
        thrust::device_vector<hipDoubleComplex*> d_ptr_A_batch(batchSize), d_ptr_rhs(batchSize);
        thrust::host_vector<hipDoubleComplex*, pinnedAllocPtr> h_ptr_A_batch(batchSize), h_ptr_rhs(batchSize);
        // Initialise shifts
        int shift_global_A, shift_batch_A, shift_global_rhs;
        shift_global_A = tid*freq_max*nnz_max;
        // Set cuBLAS stream
        hipblasSetStream(cublasHandle[tid], streams[tid]);
    // Loop over each matrix size
    #pragma omp for
        for (size_t i = 0; i < subComponents; ++i){
            /*--------------------------------------
            Update pointers to each matrix A and RHS
            --------------------------------------*/
            // Initialise Shifts
            shift_global_rhs = 0;
            shift_batch_A    = 0;
            // Loop over batch (assume batchSize = freq_max)
            for (size_t j = 0; j < batchSize; ++j){
                // Update pointers for batched operations
                h_ptr_A_batch[j] = d_ptr_A_batch_base + shift_batch_A      + shift_global_A;
                h_ptr_rhs[j]     = d_ptr_rhs_base     + shift_local_rhs[i] + shift_global_rhs;
                // Update shifts
                shift_batch_A    += nnz_sub[i];
                shift_global_rhs += row;
            }

            /*------------------------
            Solve Reduced Order System
            ------------------------*/
            PUSH_RANGE("Linear System", 5)

            // Assembly Matrices in Batch
            PUSH_RANGE("Matrix Assembly", 4)
            d_ptr_A_batch = h_ptr_A_batch;
            assembly::assembleGlobalMatrixBatched(streams[tid], thrust::raw_pointer_cast(d_ptr_A_batch.data()), d_ptr_K[i], d_ptr_M[i],
                                                  nnz_sub[i], thrust::raw_pointer_cast(freq_square.data()), (int)freq_max);
            POP_RANGE // Matrix Assembly

            // LU Decomposition
            d_ptr_A_batch = h_ptr_A_batch;
            cublas_check(hipblasZgetrfBatched(cublasHandle[tid], row_sub[i], thrust::raw_pointer_cast(d_ptr_A_batch.data()), row_sub[i], NULL, d_ptr_solverInfo, batchSize));

            // Solve x = A\b
            d_ptr_rhs = h_ptr_rhs;
            cublas_check(hipblasZgetrsBatched(cublasHandle[tid], HIPBLAS_OP_N, row_sub[i], 1, thrust::raw_pointer_cast(d_ptr_A_batch.data()), row_sub[i], NULL,
                                             thrust::raw_pointer_cast(d_ptr_rhs.data()), row_sub[i], &solverInfo_solve, batchSize));
            POP_RANGE // Linear System

            /*-----------------
            Synchronize Streams
            -----------------*/
            hipStreamSynchronize(streams[tid]);

        } // matrix loop
    } // omp parallel

    timerMOR.stop();
    std::cout << ">> Krylov Subspace Method finished" << std::endl;
    std::cout << ">>>> Time taken = " << timerMOR.getDurationMicroSec()*1e-6 << " sec" << "\n" << std::endl;
    POP_RANGE // Krylov Subspace Method

    // Copy solution and re-project matrix from device to host
    PUSH_RANGE("Solution Transfer to Host", 8)
    timerDataD2H.start();
    thrust::host_vector<hipDoubleComplex> rhs = d_rhs;
    timerDataD2H.stop();
    POP_RANGE // Solution Transfer to Host
    std::cout << ">> Solutions copied to Host" << std::endl;
    std::cout << ">>>> Time taken = " << timerDataD2H.getDurationMicroSec()*1e-6 << " sec" << "\n" << std::endl;

    // Write solutions
/*
    io::writeSolVecComplex(rhs, filepath_sol, filename_sol);
*/

    // Destroy cuBLAS & streams
    for (size_t i = 0; i < num_threads; ++i){
        hipblasDestroy(cublasHandle[i]);
        hipStreamDestroy(streams[i]);
    }

    timerTotal.stop();
    std::cout << ">>>> End of program" << std::endl;
    std::cout << ">>>>>> Total execution time (s) = " << timerTotal.getDurationMicroSec()*1e-6 << "\n" << std::endl;
}
