// Libraries
#include <iostream>
#include <vector>
#include <string>
#include <cmath>

// OpenMP
#include <omp.h>

// CUDA
#include <hip/hip_runtime.h>

// THRUST
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// CUBLAS
#include <hipblas.h>

// CUSOLVER
#include <hipsolver.h>

// Header files
#include "io/io.cuh"
#include "helper/Timer.cuh"

// Definitions
#define	PI	3.14159265359

int main (int argc, char *argv[]){

	// Vector of filepaths
	std::string filepath[2];
	filepath[0] = "/opt/software/examples/MOR/r_approx_180/\0";
	filepath[1] = "/opt/software/examples/MOR/r_approx_300/\0";

	// Solution filepath
	std::string filepath_sol = "output/";

	// Solution filename
	std::string filename_sol = "solution.dat";

	// Array of matrix sizes (row)
	int row_sub[] = {126, 132, 168, 174, 180, 186, 192, 288, 294, 300, 306, 312};

	// Array of filenames
	std::string baseName_K = "KSM_Stiffness_r\0";
	std::string baseName_M = "KSM_Mass_r\0";
	std::string baseName_D = "KSM_Damping_r\0";
	std::string base_format = ".mtx\0";
	std::string filename_K[12];
	std::string filename_M[12];
	std::string filename_D[12];

	// Parameters
	bool isComplex = 1;
	double freq, freq_square;
	double freq_min = 1;
	double freq_max = 2000;
	const double alpha = 4*PI*PI;
	hipDoubleComplex one;	// Dummy scailing factor for global matrix assembly
	one.x = 1;
	one.y = 0;
	hipDoubleComplex rhs_val;
	rhs_val.x = (double)1.0;
	rhs_val.y = (double)0.0;
	int mat_repetition = 5;

	timerTotal.start();
	// Library initialisation
	hipblasStatus_t hipblasStatus_t;
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	hipsolverStatus_t cusolverStatus = HIPSOLVER_STATUS_ALLOC_FAILED;
	hipsolverHandle_t cusolverHandle;
	cusolverStatus = hipsolverDnCreate(&cusolverHandle);

	// Create matrix host_vectors
	thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub(12);
	thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub(12);
	thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub(12);

	// Read and process MTX file
	for (size_t i = 0; i < 7; i++){
		filename_K[i] = baseName_K + std::to_string(row_sub[i]) + base_format;
		filename_M[i] = baseName_M + std::to_string(row_sub[i]) + base_format;
		filename_D[i] = baseName_D + std::to_string(row_sub[i]) + base_format;
		io::readMtxDense(K_sub[i], filepath[0], filename_K[i], isComplex);
		io::readMtxDense(M_sub[i], filepath[0], filename_M[i], isComplex);
		io::readMtxDense(D_sub[i], filepath[0], filename_D[i], isComplex);
		K_sub[i].pop_back();
		M_sub[i].pop_back();
		D_sub[i].pop_back();
	}
	for (size_t i = 7; i < 12; i++){
		filename_K[i] = baseName_K + std::to_string(row_sub[i]) + base_format;
		filename_M[i] = baseName_M + std::to_string(row_sub[i]) + base_format;
		filename_D[i] = baseName_D + std::to_string(row_sub[i]) + base_format;
		io::readMtxDense(K_sub[i], filepath[1], filename_K[i], isComplex);
		io::readMtxDense(M_sub[i], filepath[1], filename_M[i], isComplex);
		io::readMtxDense(D_sub[i], filepath[1], filename_D[i], isComplex);
		K_sub[i].pop_back();
		M_sub[i].pop_back();
		D_sub[i].pop_back();
	}
	std::cout << ">> Matrices imported" << std::endl;

	// Get matrix sizes
	thrust::host_vector<int> size_sub(12);
	int size = 0;
	int row = 0;
	for (size_t i = 0; i < 12; i++){
		size_sub[i] = row_sub[i]*row_sub[i];
		size += size_sub[i];
		row  += row_sub[i];
	}
	row *= mat_repetition;
	size *= mat_repetition;

	// Combine matrices into a single array on host (to make use of GPU's high bandwidth. We could also import the matrices directly like this)
	thrust::host_vector<hipDoubleComplex> K(size);
	thrust::host_vector<hipDoubleComplex> M(size);
	thrust::host_vector<hipDoubleComplex> D(size);
	int  array_shift = 0;
	auto K_sub_ptr = &K_sub[0];
	auto M_sub_ptr = &M_sub[0];
	auto D_sub_ptr = &D_sub[0];
	for (size_t j = 0; j < mat_repetition; j++){
		for (size_t i = 0; i < 12; i++){
			K_sub_ptr = &K_sub[i];
			M_sub_ptr = &M_sub[i];
			D_sub_ptr = &D_sub[i];
			thrust::copy(K_sub_ptr->begin(), K_sub_ptr->end(), K.begin()+array_shift);
			thrust::copy(M_sub_ptr->begin(), M_sub_ptr->end(), M.begin()+array_shift);
			thrust::copy(D_sub_ptr->begin(), D_sub_ptr->end(), D.begin()+array_shift);
			array_shift += size_sub[i];
		}
	}
	std::cout <<">> Matrices combined" << std::endl;

	// Send matrices to device
	timerMatrixCpy.start();
	thrust::device_vector<hipDoubleComplex> d_K = K;
	thrust::device_vector<hipDoubleComplex> d_M = M;
	thrust::device_vector<hipDoubleComplex> d_D = D;
	timerMatrixCpy.stop();
	std::cout << ">> Matrices copied to device " << std::endl;
	std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

	// Create RHS directly on device
	timerMatrixCpy.start();
	thrust::device_vector<hipDoubleComplex> d_rhs(row, rhs_val);
	thrust::device_vector<hipDoubleComplex> d_rhs_buf = d_rhs;
	timerMatrixCpy.stop();
	std::cout << ">> RHS copied to device " << std::endl;
	std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

	// Create matrix device_vectors
	thrust::device_vector<hipDoubleComplex> d_A(size);

	// Get raw pointers to matrices
	hipDoubleComplex *d_ptr_K = thrust::raw_pointer_cast(d_K.data());
	hipDoubleComplex *d_ptr_M = thrust::raw_pointer_cast(d_M.data());
	hipDoubleComplex *d_ptr_D = thrust::raw_pointer_cast(d_D.data());
	hipDoubleComplex *d_ptr_A = thrust::raw_pointer_cast(d_A.data());

	// Get raw pointers to RHS vectors
	hipDoubleComplex *d_ptr_rhs = thrust::raw_pointer_cast(d_rhs.data());

	// Create solution vector on host
	thrust::host_vector<hipDoubleComplex> sol(row*freq_max);

	// Create solution vector on device
	thrust::device_vector<hipDoubleComplex> d_sol(row*freq_max);

	timerMatrixComp.start();
	// M = 4*pi^2*M (Single computation suffices)
	hipblasStatus_t = hipblasZdscal(cublasHandle, size, &alpha, d_ptr_M, 1);
	timerMatrixComp.stop();
	std::cout << ">> M_tilde computed with cuBLAS" << std::endl;
	std::cout << ">>>> Time taken = " << timerMatrixComp.getDurationMicroSec()*1e-6 << " (sec)\n" << std::endl;
	if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS){
		std::cout << "cublas failed!" << std::endl;
	}

	// LU decomposition prep
	thrust::host_vector<int> solverInfo(1);
	thrust::device_vector<int> d_solverInfo(1);
	int *d_ptr_solverInfo = thrust::raw_pointer_cast(d_solverInfo.data());

/*
	// Pivots
	thrust::device_vector<int> d_pivot(row);
	thrust::sequence(d_pivot.begin(), d_pivot.end()-(int)row/2, row-1, -1);
	int *d_ptr_pivot = thrust::raw_pointer_cast(d_pivot.data());
*/

	// Compute workspace size
	int totalSizeWorkspace = 0;
	thrust::host_vector<int> sizeWorkspace(12*mat_repetition);
	auto sizeWorkspace_ptr = &sizeWorkspace[0];
	for (size_t j = 0; j < mat_repetition; j++){
		for (size_t i = 0; i < 12; i++){
			sizeWorkspace_ptr = &sizeWorkspace[i+12*j];
			cusolverStatus = hipsolverDnZgetrf_bufferSize(cusolverHandle, row_sub[i], row_sub[i], d_ptr_A, row_sub[i], sizeWorkspace_ptr);
			if (cusolverStatus != HIPSOLVER_STATUS_SUCCESS) std::cout << ">> cuSolver workspace size computation failed\n" << std::endl;
			totalSizeWorkspace += sizeWorkspace[i];
		}
	}

	// Create workspace
	thrust::device_vector<hipDoubleComplex> d_workspace(totalSizeWorkspace);
	hipDoubleComplex *d_ptr_workspace = thrust::raw_pointer_cast(d_workspace.data());

	timerLoop.start();
	int sol_shift = 0;
	// Loop over frequency
	for (size_t it = (size_t)freq_min; it <= (size_t)freq_max; it++){
		timerIteration.start();

		// Compute scaling
		freq = (double)it;
		freq_square = -(freq*freq);

		// Assemble global matrix ( A = K - f^2*M_tilde )
		d_A = d_M;
		// Scale A with -f^2
		hipblasStatus_t = hipblasZdscal(cublasHandle, size, &freq_square, d_ptr_A, 1);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS){
			std::cout << "cublas failed!" << std::endl;
		}
		// Sum A with K
		hipblasStatus_t = hipblasZaxpy(cublasHandle, size, &one, d_ptr_K, 1, d_ptr_A, 1);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS){
			std::cout << "cublas failed during matrix assembly!" << std::endl;
		}

		array_shift = 0;
		size_t row_shift = 0;
		size_t workspace_shift = 0;
		for (size_t j = 0; j < mat_repetition; j++){
			for (size_t i = 0; i < 12; i++){
				// LU decomposition
				cusolverStatus = hipsolverDnZgetrf(cusolverHandle, row_sub[i], row_sub[i], d_ptr_A + array_shift, row_sub[i], d_ptr_workspace + workspace_shift, NULL, d_ptr_solverInfo);
				if (cusolverStatus != HIPSOLVER_STATUS_SUCCESS) std::cout << ">> cuSolver LU decomposition failed" << std::endl;
				solverInfo = d_solverInfo;
				if (solverInfo[0] != 0){
					std::cout << ">>>> LU decomposition failed" << std::endl;
					std::cout << ">>>> solverInfo = " << solverInfo[0] << std::endl;
				}

				// Solve x = A\b
				cusolverStatus = hipsolverDnZgetrs(cusolverHandle, HIPBLAS_OP_N, row_sub[i], 1, d_ptr_A + array_shift, row_sub[i], NULL, d_ptr_rhs + row_shift, row_sub[i], d_ptr_solverInfo);
				if (cusolverStatus != HIPSOLVER_STATUS_SUCCESS) std::cout << ">> System couldn't be solved" << std::endl;
				solverInfo = d_solverInfo;
				if (solverInfo[0] != 0) {
					std::cout << ">>>> System solution failure" << std::endl;
				}
				array_shift += size_sub[i];
				row_shift += row_sub[i];
				workspace_shift += sizeWorkspace[i];
			}
		}
		// Copy the solution to solution vector
		thrust::copy(d_rhs.begin(), d_rhs.end(), d_sol.begin() + sol_shift);
		sol_shift += row;
		// Reset RHS
		d_rhs = d_rhs_buf;
	}
	timerLoop.stop();

	std::cout << ">>>> Frequency loop finished" << std::endl;
	std::cout << ">>>>>> Time taken (s) = " << timerLoop.getDurationMicroSec()*1e-6 << "\n" << std::endl;

	sol = d_sol;
	// Write out solution vectors
	io::writeSolVecComplex(sol, filepath_sol, filename_sol);

	// Destroy cuBLAS & cuSolver
	hipblasDestroy(cublasHandle);
	hipsolverDnDestroy(cusolverHandle);

	timerTotal.stop();
	std::cout << ">>>>>> Total execution time (s) = " << timerTotal.getDurationMicroSec()*1e-6 << "\n" << std::endl;
}
