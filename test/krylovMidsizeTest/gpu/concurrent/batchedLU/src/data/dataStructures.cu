// Libraries
#include <string>
#include <thrust/host_vector.h>

// Header Files
#include "dataStructures.cuh"
#include "../io/io.cuh"

// Namespace
using namespace staccato;

void data::getInfoHostDataStructure(
                                    thrust::host_vector<int> &shift_local_A, thrust::host_vector<int> &shift_local_rhs,
                                    thrust::host_vector<int> &row_sub, thrust::host_vector<int> &nnz_sub,
                                    int &nnz, int &row, int &nnz_max, int mat_repetition, int row_baseline[]
                                   )
{
    // Get matrix sizes and local shifts
    nnz = 0;
    row = 0;
    size_t idx;
    int mat_shift   = 0;
    int sol_shift   = 0;
    for (size_t j = 0; j < mat_repetition; ++j){
        for (size_t i = 0; i < 12; ++i){
            // Index for combined matrix
            idx = i + 12*j;
            // Sub-component matrix & vector sizes
            row_sub[idx]       = row_baseline[i];
            nnz_sub[idx]       = row_sub[i]*row_sub[i];
            // Accumulate total matrix & vector sizes
            nnz   += nnz_sub[idx];
            row   += row_sub[idx];
            // (Local) shifts for each sub-components from combined matrix
            shift_local_A[idx]   = mat_shift;
            shift_local_rhs[idx] = sol_shift;
            // Update shifts
            mat_shift   += nnz_sub[idx];
            sol_shift   += row_sub[idx];
        }
    }
    // Get maximum matrix size
    auto nnz_max_it   = thrust::max_element(nnz_sub.begin(), nnz_sub.end());
    nnz_max   = *nnz_max_it;
}

void data::getInfoDeviceDataStructure(
                                      thrust::device_vector<hipDoubleComplex*> &d_ptr_K,
                                      thrust::device_vector<hipDoubleComplex*> &d_ptr_M,
                                      thrust::device_vector<hipDoubleComplex*> &d_ptr_D,
                                      hipDoubleComplex *d_ptr_K_base,
                                      hipDoubleComplex *d_ptr_M_base,
                                      hipDoubleComplex *d_ptr_D_base,
                                      thrust::host_vector<int> nnz_sub,
                                      int subComponents
                                     )
{
    // Get pointers to each sub-components in combined matrices on device
    int mat_shift   = 0;
    for (size_t i = 0; i < subComponents; ++i){
        d_ptr_K[i] = d_ptr_K_base + mat_shift;
        d_ptr_M[i] = d_ptr_M_base + mat_shift;
        d_ptr_D[i] = d_ptr_D_base + mat_shift;
        mat_shift   += nnz_sub[i];
    }
}

void data::combineHostMatrices(
                               thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub,
                               thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub,
                               thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub,
                               thrust::host_vector<hipDoubleComplex> &K, thrust::host_vector<hipDoubleComplex> &M, thrust::host_vector<hipDoubleComplex> &D,
                               int nnz, int mat_repetition, thrust::host_vector<int> nnz_sub
                              )
{
    K.resize(nnz);
    M.resize(nnz);
    D.resize(nnz);
    // Combine matrices into a single array
    auto K_sub_ptr = &K_sub[0];
    auto M_sub_ptr = &M_sub[0];
    auto D_sub_ptr = &D_sub[0];
    size_t array_shift = 0;
    for (size_t j = 0; j < mat_repetition; ++j){
        for (size_t i = 0; i < 12; ++i){
            K_sub_ptr = &K_sub[i];
            M_sub_ptr = &M_sub[i];
            D_sub_ptr = &D_sub[i];
            thrust::copy(K_sub_ptr->begin(), K_sub_ptr->end(), K.begin() + array_shift);
            thrust::copy(M_sub_ptr->begin(), M_sub_ptr->end(), M.begin() + array_shift);
            thrust::copy(D_sub_ptr->begin(), D_sub_ptr->end(), D.begin() + array_shift);
            array_shift   += nnz_sub[i];
        }
    }
}

void data::constructHostDataStructure(
                                      std::string filename_K[], std::string filename_M[], std::string filename_D[],
                                      std::string filepath[],
                                      std::string baseName_K,   std::string baseName_M,   std::string baseName_D, std::string base_format,
                                      int row_baseline[],
                                      thrust::host_vector<thrust::host_vector<hipDoubleComplex>> &K_sub,
                                      thrust::host_vector<thrust::host_vector<hipDoubleComplex>> &M_sub,
                                      thrust::host_vector<thrust::host_vector<hipDoubleComplex>> &D_sub
                                     )
{
    /*------------------------
    READ AND PROCESS MTX FILES
    ------------------------*/
    for (size_t i = 0; i < 7; ++i){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[0], filename_K[i], true);
        io::readMtxDense(M_sub[i], filepath[0], filename_M[i], true);
        io::readMtxDense(D_sub[i], filepath[0], filename_D[i], true);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }
    for (size_t i = 7; i < 12; ++i){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[1], filename_K[i], true);
        io::readMtxDense(M_sub[i], filepath[1], filename_M[i], true);
        io::readMtxDense(D_sub[i], filepath[1], filename_D[i], true);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }
}
