#include "hip/hip_runtime.h"
// Libraries
#include <iostream>
#include <vector>
#include <string>
#include <cmath>
#include <algorithm>

// OpenMP
#include <omp.h>

// THRUST
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// CUBLAS
#include <hipblas.h>

// CUSPARSE
#include <hipsparse.h>

// Header files
#include "io/io.cuh"
#include "helper/Timer.cuh"
#include "helper/helper.cuh"
#include "helper/math.cuh"

// Definitions
#define	PI	3.14159265359

int main (int argc, char *argv[]){

    // Command line arguments
    if (argc < 5){
        std::cerr << ">> Usage: " << argv[0] << " -f <maximum frequency> -m <matrix repetition> -stream <number of CUDA streams>" << std::endl;
        std::cerr << ">> NOTE: There are 12 matrices and matrix repetition increases the total number of matrices (e.g. matrix repetition of 5 will use 60 matrices)" << std::endl;
        std::cerr << "         Frequency starts from 1 to maximum frequency" << std::endl;
        std::cerr << "         Default number of CUDA streams is 1" << std::endl;
        std::cerr << "         Ratio of maximum frequency to number of CUDA streams must be an integer" << std::endl;
        return 1;
    }

    double freq_max = atof(argv[2]);
    int mat_repetition = atoi(argv[4]);
    int num_matrix = mat_repetition*12;
    int num_streams = 1;

    if (argc > 6) num_streams = atoi(argv[6]);
    int num_threads = num_streams;
    std::cout << ">> Maximum Frequency: " << freq_max << std::endl;
    std::cout << ">> Total number of matrices: " << num_matrix << std::endl;
    std::cout << ">> Number of CUDA streams: " << num_streams << "\n" << std::endl;

    if (((int)freq_max % num_streams) != 0) {
        std::cerr << ">> ERROR: Invalid number of streams\n" << std::endl;
        return 1;
    }

    // Vector of filepaths
    std::string filepath[2];
    filepath[0] = "/opt/software/examples/MOR/r_approx_180/\0";
    filepath[1] = "/opt/software/examples/MOR/r_approx_300/\0";

    // Solution filepath
    std::string filepath_sol = "output/";

    // Solution filename
    std::string filename_sol = "solution.dat";

    // Array of matrix sizes (row)
    int row_baseline[] = {126, 132, 168, 174, 180, 186, 192, 288, 294, 300, 306, 312};

    // Array of filenames
    std::string baseName_K = "KSM_Stiffness_r\0";
    std::string baseName_M = "KSM_Mass_r\0";
    std::string baseName_D = "KSM_Damping_r\0";
    std::string base_format = ".mtx\0";
    std::string filename_K[12];
    std::string filename_M[12];
    std::string filename_D[12];

    // Parameters
    bool isComplex = 1;
    thrust::host_vector<double> freq(num_streams);
    thrust::host_vector<double> freq_square(num_streams);
    double freq_min = 1;
    const double alpha = 4*PI*PI;
    hipDoubleComplex one;			// Dummy scailing factor for global matrix assembly
    one.x = 1;
    one.y = 0;
    hipDoubleComplex rhs_val;
    rhs_val.x = (double)1.0;
    rhs_val.y = (double)0.0;

    // OpenMP
    int tid;
    omp_set_num_threads(num_threads);

    timerTotal.start();

    // Library initialisation
    hipblasStatus_t hipblasStatus_t;
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    hipsparseStatus_t cusparseStatus;
    hipsparseHandle_t cusparseHandle;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    // Create matrix host_vectors
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub(12);
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub(12);
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub(12);

    // Read and process MTX file
    for (size_t i = 0; i < 7; i++){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[0], filename_K[i], isComplex);
        io::readMtxDense(M_sub[i], filepath[0], filename_M[i], isComplex);
        io::readMtxDense(D_sub[i], filepath[0], filename_D[i], isComplex);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }

    for (size_t i = 7; i < 12; i++){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[1], filename_K[i], isComplex);
        io::readMtxDense(M_sub[i], filepath[1], filename_M[i], isComplex);
        io::readMtxDense(D_sub[i], filepath[1], filename_D[i], isComplex);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }
    std::cout << ">> Matrices imported" << std::endl;

    // Get matrix sizes
    thrust::host_vector<int> row_sub(num_matrix);
    thrust::host_vector<int> size_sub(num_matrix);
    int nnz = 0;
    int row = 0;
    size_t idx;
    for (size_t j = 0; j < mat_repetition; j++){
        for (size_t i = 0; i < 12; i++){
            idx = i + 12*j;
            row_sub[idx] = row_baseline[i];
            size_sub[idx] = row_sub[i]*row_sub[i];
            nnz += size_sub[idx];
            row  += row_sub[idx];
        }
    }

    // Combine matrices into a single array
    thrust::host_vector<hipDoubleComplex> K(nnz);
    thrust::host_vector<hipDoubleComplex> M(nnz);
    thrust::host_vector<hipDoubleComplex> D(nnz);
    auto K_sub_ptr = &K_sub[0];
    auto M_sub_ptr = &M_sub[0];
    auto D_sub_ptr = &D_sub[0];
    size_t array_shift = 0;
    for (size_t j = 0; j < mat_repetition; j++){
        for (size_t i = 0; i < 12; i++){
            K_sub_ptr = &K_sub[i];
            M_sub_ptr = &M_sub[i];
            D_sub_ptr = &D_sub[i];
            thrust::copy(K_sub_ptr->begin(), K_sub_ptr->end(), K.begin() + array_shift);
            thrust::copy(M_sub_ptr->begin(), M_sub_ptr->end(), M.begin() + array_shift);
            thrust::copy(D_sub_ptr->begin(), D_sub_ptr->end(), D.begin() + array_shift);
            array_shift += size_sub[i];
        }
    }

    std::cout <<">> Matrices combined\n" << std::endl;

    // Generate CSR format
    timerAux.start();
    thrust::host_vector<int> csrRowPtr(row+1);
    thrust::host_vector<int> csrColInd(nnz);
    generateCSR(csrRowPtr, csrColInd, row_sub, size_sub, row, nnz, num_matrix);
    thrust::device_vector<int> d_csrRowPtr = csrRowPtr;
    thrust::device_vector<int> d_csrColInd = csrColInd;
    timerAux.stop();
    std::cout <<">> CSR Format Generated" << std::endl;
    std::cout <<">>>> Time taken = " << timerAux.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

    // Send matrices to device
    timerMatrixCpy.start();
    thrust::device_vector<hipDoubleComplex> d_K = K;
    thrust::device_vector<hipDoubleComplex> d_M = M;
    thrust::device_vector<hipDoubleComplex> d_D = D;

    timerMatrixCpy.stop();
    std::cout << ">> Matrices copied to device " << std::endl;
    std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

    // Create RHS directly on device
    timerMatrixCpy.start();
    thrust::device_vector<hipDoubleComplex> d_rhs(row, rhs_val);
    //thrust::device_vector<hipDoubleComplex> d_rhs_buf = d_rhs;
    timerMatrixCpy.stop();
    std::cout << ">> RHS copied to device " << std::endl;
    std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

    // Create assembled matrix on device
    thrust::device_vector<hipDoubleComplex> d_A(nnz*num_streams);

    // Get raw pointers to matrices
    hipDoubleComplex *d_ptr_K  = thrust::raw_pointer_cast(d_K.data());
    hipDoubleComplex *d_ptr_M  = thrust::raw_pointer_cast(d_M.data());
    hipDoubleComplex *d_ptr_D  = thrust::raw_pointer_cast(d_D.data());

    // Vector of raw pointers to assembled matrix
    thrust::host_vector<hipDoubleComplex*> d_ptr_A(num_streams);
    size_t mat_shift = 0;
    hipDoubleComplex *d_ptr_A_base = thrust::raw_pointer_cast(d_A.data());
    for (size_t i = 0; i < num_streams; i++) {
        d_ptr_A[i] = d_ptr_A_base + mat_shift;
        mat_shift += nnz;
    }

    // Get raw pointers to CSR arrays
    int *d_ptr_csrRowPtr = thrust::raw_pointer_cast(d_csrRowPtr.data());
    int *d_ptr_csrColInd = thrust::raw_pointer_cast(d_csrColInd.data());

    // Get raw pointers to RHS vectors
    hipDoubleComplex *d_ptr_rhs = thrust::raw_pointer_cast(d_rhs.data());

    // Create solution vector on host
    thrust::host_vector<hipDoubleComplex> sol(row*freq_max);

    // Create solution vector on device
    thrust::device_vector<hipDoubleComplex> d_z(row*num_streams);	// Intermediate solution
    thrust::device_vector<hipDoubleComplex> d_sol(row*freq_max);		// Final solution

    // Get raw pointers to solution vector
    thrust::host_vector<hipDoubleComplex*> d_ptr_z(num_streams);
    size_t z_shift = 0;
    for (size_t i = 0; i < num_streams; i++){
        d_ptr_z[i] = thrust::raw_pointer_cast(d_z.data() + z_shift);
        z_shift += row;
    }
    thrust::host_vector<hipDoubleComplex*> d_ptr_sol(freq_max);
    size_t sol_shift = 0;
    for (size_t i = 0; i < freq_max; i++){
        d_ptr_sol[i] = thrust::raw_pointer_cast(d_sol.data() + sol_shift);
        sol_shift += row;
    }

    timerMatrixComp.start();
    // M = 4*pi^2*M (Single computation suffices)
    hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &alpha, d_ptr_M, 1);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);
    timerMatrixComp.stop();
    std::cout << ">> M_tilde computed with cuBLAS" << std::endl;
    std::cout << ">>>> Time taken = " << timerMatrixComp.getDurationMicroSec()*1e-6 << " (sec)\n" << std::endl;

    // Stream initialisation
    hipStream_t streams[num_streams];
    for (size_t i = 0; i < num_streams; i++) hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);

    /*-----------------------------
    LU Decomposition initialisation
    -----------------------------*/
    timerAux.start();
    // Matrix Descriptions
    hipsparseMatDescr_t descr_A, descr_L, descr_U;
    hipsparseCreateMatDescr(&descr_A);
    hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);
    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    // Solver Infos
    csrilu02Info_t solverInfo_A;
    csrsv2Info_t solverInfo_L, solverInfo_U;
    hipsparseCreateCsrilu02Info(&solverInfo_A);
    hipsparseCreateCsrsv2Info(&solverInfo_L);
    hipsparseCreateCsrsv2Info(&solverInfo_U);
    // Transpose operations
    const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_U = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    // Solver policies
    const hipsparseSolvePolicy_t policy_A = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    // Zero Pivoting
    int structural_zero, numerical_zero;
    // Buffer space
    int bufferSize_A, bufferSize_L, bufferSize_U, bufferSize;
    cusparseStatus = hipsparseZcsrilu02_bufferSize(cusparseHandle, row, nnz, descr_A, d_ptr_A[0], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_A, &bufferSize_A);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
    cusparseStatus = hipsparseZcsrsv2_bufferSize(cusparseHandle, trans_L, row, nnz, descr_L, d_ptr_A[0], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_L, &bufferSize_L);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
    cusparseStatus = hipsparseZcsrsv2_bufferSize(cusparseHandle, trans_U, row, nnz, descr_U, d_ptr_A[0], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_U, &bufferSize_U);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
    bufferSize = std::max(bufferSize_A, std::max(bufferSize_L, bufferSize_U));
    thrust::device_vector<int> d_buffer(bufferSize*num_streams);
    void *d_ptr_buffer = thrust::raw_pointer_cast(d_buffer.data());
    void *d_ptr_buffer_stream;
    // Perform analysis
    cusparseStatus = hipsparseZcsrilu02_analysis(cusparseHandle, row, nnz, descr_A, d_ptr_A[0], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_A, policy_A, d_ptr_buffer);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
    cusparseStatus = hipsparseXcsrilu02_zeroPivot(cusparseHandle, solverInfo_A, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    cusparseStatus = hipsparseZcsrsv2_analysis(cusparseHandle, trans_L, row, nnz, descr_L, d_ptr_A[0], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_L, policy_L, d_ptr_buffer);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
    cusparseStatus = hipsparseZcsrsv2_analysis(cusparseHandle, trans_U, row, nnz, descr_U, d_ptr_A[0], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_U, policy_U, d_ptr_buffer);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
    timerAux.stop();
    std::cout << ">> LU decomposition initialised" << std::endl;
    std::cout << ">>>> Time taken (s) = " << timerAux.getDurationMicroSec()*1e-6 << "\n" << std::endl;

    /*------------
    Frequency Loop
    ------------*/
    std::cout << ">>>> Frequency loop started" << std::endl;
    timerLoop.start();
    for (size_t it = (size_t)freq_min; it <= (size_t)freq_max; it += num_streams){

        /*--------------------------------------------
        Assemble global matrix ( A = K - f^2*M_tilde )
        --------------------------------------------*/
        for (size_t i = 0; i < num_streams; i++){
            // Compute scaling
            freq[i] = (double)it + i;
            freq_square[i] = -(freq[i]*freq[i]);
            // Copy M to A
            thrust::copy(d_M.begin(), d_M.end(), d_ptr_A[i]);
            // Scale A with -f^2
            hipblasSetStream(cublasHandle, streams[i]);
            hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &freq_square[i], d_ptr_A[i], 1);
            assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);
        }

/*
        for (size_t i = 0; i < num_streams; i++){
            // Sum A with K
            hipblasSetStream(cublasHandle, streams[i]);
            hipblasStatus_t = hipblasZaxpy(cublasHandle, nnz, &one, d_ptr_K, 1, d_ptr_A[i], 1);
            assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);
        }
*/

        /*--------------
        LU Decomposition
        --------------*/
/*
        for (size_t i = 0; i < num_streams; i++){
            d_ptr_buffer_stream = (void*)((int*)d_ptr_buffer+i*bufferSize);
            hipsparseSetStream(cusparseHandle, streams[i]);
            cusparseStatus = hipsparseZcsrilu02(cusparseHandle, row, nnz, descr_A, d_ptr_A[i], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_A, policy_A, d_ptr_buffer_stream);
            assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
        }
*/
/*
        for (size_t i = 0; i < num_streams; i++){
            hipsparseSetStream(cusparseHandle, streams[i]);
            cusparseStatus = hipsparseXcsrilu02_zeroPivot(cusparseHandle, solverInfo_A, &numerical_zero);
            if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
        }
*/

        /*-----------
        Solve x = A\b
        -----------*/
/*
        for (size_t i = 0; i < num_streams; i++){
            size_t freq_idx = freq[i]-1;
            // Solve z = L\b
            hipsparseSetStream(cusparseHandle, streams[i]);
            cusparseStatus = hipsparseZcsrsv2_solve(cusparseHandle, trans_L, row, nnz, &one, descr_L, d_ptr_A[i], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_L,
                    d_ptr_rhs, d_ptr_z[i], policy_L, d_ptr_buffer_stream);
            assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
            // Solve x = U\z
            hipsparseSetStream(cusparseHandle, streams[i]);
            cusparseStatus = hipsparseZcsrsv2_solve(cusparseHandle, trans_U, row, nnz, &one, descr_U, d_ptr_A[i], d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_U,
                    d_ptr_z[i], d_ptr_sol[freq_idx], policy_U, d_ptr_buffer_stream);
            assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
        }
        // Synchronize streams
        for (size_t i = 0; i < num_streams; i++) hipStreamSynchronize(streams[i]);
*/
    }
    timerLoop.stop();

    std::cout << ">>>> Frequency loop finished" << std::endl;
    std::cout << ">>>>>> Time taken (s) = " << timerLoop.getDurationMicroSec()*1e-6 << "\n" << std::endl;

    sol = d_sol;

    // Write out solution vectors
    //io::writeSolVecComplex(sol, filepath_sol, filename_sol);

    // Destroy cuBLAS & cuSparse
    hipblasDestroy(cublasHandle);
    hipsparseDestroy(cusparseHandle);

    // Destroy streams
    for (size_t i = 0; i < num_streams; i++) hipStreamDestroy(streams[i]);

    timerTotal.stop();
    std::cout << ">>>>>> Total execution time (s) = " << timerTotal.getDurationMicroSec()*1e-6 << "\n" << std::endl;
}
