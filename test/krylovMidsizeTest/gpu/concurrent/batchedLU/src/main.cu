// Libraries
#include <iostream>
#include <string>
#include <cmath>

// OpenMP
#include <omp.h>

// THRUST
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// CUBLAS
#include <hipblas.h>

// NVTX: https://devblogs.nvidia.com/cuda-pro-tip-generate-custom-application-profile-timelines-nvtx/
#include <nvToolsExt.h>

// Header files
#include "config/config.cuh"
#include "io/io.cuh"
#include "data/dataStructures.cuh"
#include "solver/assembly.cuh"
#include "helper/Timer.cuh"
#include "helper/helper.cuh"

// Definitions
#define	PI	3.14159265359
#define MAX_NUM_THREADS 32

// Pinned Allocators
typedef thrust::system::cuda::experimental::pinned_allocator<int> pinnedAllocInt;
typedef thrust::system::cuda::experimental::pinned_allocator<hipDoubleComplex> pinnedAlloc;
typedef thrust::system::cuda::experimental::pinned_allocator<hipDoubleComplex*> pinnedAllocPtr;

// Namespace
using namespace staccato;

int main (int argc, char *argv[]){

    nvtxRangePushA("Initial Configuration (Host)");
    /*--------------------
    COMMAND LINE ARGUMENTS
    --------------------*/
    double freq_max;
    int mat_repetition, subComponents, num_streams, num_threads, batchSize;
    // Configure test environment with command line arguments
    config::configureTest(argc, argv, freq_max, mat_repetition, subComponents, num_streams, num_threads, batchSize);

    /*---------------------
    FILEPATHS AND FILENAMES
    ---------------------*/
    // Vector of filepaths
    std::string filepath[2];
    filepath[0] = "/opt/software/examples/MOR/r_approx_180/\0";
    filepath[1] = "/opt/software/examples/MOR/r_approx_300/\0";
    // Solution filepath
    std::string filepath_sol = "output/";
    // Solution filename
    std::string filename_sol = "solution.dat";
    // Array of filenames
    std::string baseName_K = "KSM_Stiffness_r\0";
    std::string baseName_M = "KSM_Mass_r\0";
    std::string baseName_D = "KSM_Damping_r\0";
    std::string base_format = ".mtx\0";
    std::string filename_K[12];
    std::string filename_M[12];
    std::string filename_D[12];
    std::string filename_input[12];

    /*--------
    PARAMETERS
    --------*/
    const double alpha = 4*PI*PI;
    hipDoubleComplex rhs_val;
    rhs_val.x = (double)1.0;
    rhs_val.y = (double)0.0;
    // Array of matrix sizes (row)
    int row_baseline[] = {126, 132, 168, 174, 180, 186, 192, 288, 294, 300, 306, 312};
    // Frequency vector
    thrust::host_vector<int, pinnedAllocInt> freq(batchSize);
    thrust::host_vector<int, pinnedAllocInt> freq_square(batchSize);

    /*----------------------------
    OPENMP & CUBLAS INITIALIZATION
    ----------------------------*/
    // OpenMP
    int tid;
    omp_set_num_threads(num_threads);
    // cuBLAS
    timerTotal.start();
    hipblasHandle_t cublasHandle[MAX_NUM_THREADS];
    for (size_t i = 0; i < num_threads; ++i) hipblasCreate(cublasHandle + i);

    /*-----------------------
    CHECK MEMORY REQUIREMENTS
    -----------------------*/
    config::check_memory(mat_repetition, freq_max, num_threads);

    nvtxRangePop(); // Initial Configuration

    /*--------------------
    DATA STRUCTURES (HOST)
    --------------------*/
    nvtxRangePushA("Data Structures (Host)");
    // Create matrix host_vectors
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub(12);
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub(12);
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub(12);
    thrust::host_vector<hipDoubleComplex> K;
    thrust::host_vector<hipDoubleComplex> M;
    thrust::host_vector<hipDoubleComplex> D;
    // Array information
    thrust::host_vector<int> row_sub(subComponents);
    thrust::host_vector<int> nnz_sub(subComponents);
    int nnz, row, nnz_max;
    thrust::host_vector<int> shift_local_A(subComponents);
    thrust::host_vector<int> shift_local_rhs(subComponents);
    // Set up host data structures
    data::constructHostDataStructure(filename_K, filename_M, filename_D, filepath, baseName_K, baseName_M, baseName_D, base_format, row_baseline,
                                     K_sub, M_sub, D_sub, shift_local_A, shift_local_rhs, row_sub, nnz_sub, nnz, row, nnz_max, mat_repetition, K, M, D);

    nvtxRangePop(); // Data Structures (Host)

    /*----------------------
    DATA STRUCTURES (DEVICE)
    ----------------------*/
    nvtxRangePushA("Data Structures (Device)");
    // Send matrices to device
    thrust::device_vector<hipDoubleComplex> d_K = K;
    thrust::device_vector<hipDoubleComplex> d_M = M;
    thrust::device_vector<hipDoubleComplex> d_D = D;
    // Create RHS vector directly on device (will be replaced with send operation)
    thrust::device_vector<hipDoubleComplex> d_rhs(row*freq_max, rhs_val);
    // Create matrix device_vectors
    thrust::device_vector<hipDoubleComplex> d_A(num_threads*freq_max*nnz_max);
    // Get raw pointers to device matrices & vectors
    hipDoubleComplex *d_ptr_K_base = thrust::raw_pointer_cast(d_K.data());
    hipDoubleComplex *d_ptr_M_base = thrust::raw_pointer_cast(d_M.data());
    hipDoubleComplex *d_ptr_D_base = thrust::raw_pointer_cast(d_D.data());
    hipDoubleComplex *d_ptr_A_base = thrust::raw_pointer_cast(d_A.data());
    hipDoubleComplex *d_ptr_rhs_base = thrust::raw_pointer_cast(d_rhs.data());
    // Create array of pointers for each sub-components from combined matrices on device
    thrust::host_vector<hipDoubleComplex*> h_ptr_K(subComponents);
    thrust::host_vector<hipDoubleComplex*> h_ptr_M(subComponents);
    thrust::host_vector<hipDoubleComplex*> h_ptr_D(subComponents);
    // Get information from device data structures
    data::getInfoDeviceDataStructure(h_ptr_K, h_ptr_M, h_ptr_D, d_ptr_K_base, d_ptr_M_base, d_ptr_D_base, nnz_sub, subComponents);

    nvtxRangePop(); // Data Structures (Device)


    /*--------------------------------
    Krylov Subspace Method Preparation
    --------------------------------*/
    nvtxRangePushA("Krylov Subspace Method Preparation");
    // M = 4*pi^2*M
    cublas_check(hipblasZdscal(cublasHandle[0], nnz, &alpha, d_ptr_M_base, 1));
    // Solver Info for batched LU decomposition
    thrust::device_vector<int> d_solverInfo(batchSize);
    int *d_ptr_solverInfo = thrust::raw_pointer_cast(d_solverInfo.data());
    int solverInfo_solve;
    // Stream initialisation
    hipStream_t streams[num_streams];
    for (size_t i = 0; i < num_streams; ++i){
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
        std::cout << ">> Stream " << i << " created" << std::endl;
    }

    nvtxRangePop(); // Krylov Subspace Method Preparation

    /*--------------------
    Krylov Subspace Method
    --------------------*/
    nvtxRangePushA("Krylov Subspace Method");
    timerLoop.start();
    std::cout << "\n>> Matrix loop started for batched execution" << std::endl;
#pragma omp parallel private(tid) num_threads(num_threads)
    {
        // Get thread number
        tid = omp_get_thread_num();
        // Allocate vector of array pointers to A in each thread
        thrust::device_vector<hipDoubleComplex*> d_ptr_A(batchSize);
        thrust::device_vector<hipDoubleComplex*> d_ptr_rhs(batchSize);
        thrust::host_vector<hipDoubleComplex*, pinnedAllocPtr> h_ptr_A(batchSize);
        thrust::host_vector<hipDoubleComplex*, pinnedAllocPtr> h_ptr_rhs(batchSize);
        // Initialise shifts
        int shift_global_A, shift_batch_A, shift_global_rhs;
        shift_global_A = tid*freq_max*nnz_max;
        // Set cuBLAS stream
        hipblasSetStream(cublasHandle[tid], streams[tid]);
    // Loop over each matrix size
    #pragma omp for
        for (size_t i = 0; i < subComponents; ++i){
            /*---------------------------------------------------------------
            Assemble Global Matrix & Update pointers to each matrix A and RHS
            ---------------------------------------------------------------*/
            // Initialise Shifts
            shift_global_rhs = 0;
            shift_batch_A = 0;
            // Loop over batch (assume batchSize = freq_max)
            for (size_t j = 0; j < batchSize; ++j){
                // Update matrix A pointer
                h_ptr_A[j] = d_ptr_A_base + shift_batch_A + shift_global_A;
                // Update rhs pointer
                h_ptr_rhs[j] = d_ptr_rhs_base + shift_local_rhs[i] + shift_global_rhs;
                // Compute frequency (assume batchSize = freq_max)
                freq[j] = (j+1);
                freq_square[j] = -(freq[j]*freq[j]);
                // Assemble matrix
                nvtxRangePushA("Matrix Assembly");
                assembly::assembleGlobalMatrixBatched(streams[tid], h_ptr_A[j], h_ptr_K[i], h_ptr_M[i], nnz_sub[i], freq_square[j]);
                nvtxRangePop();
                // Update shifts
                shift_batch_A    += nnz_sub[i];
                shift_global_rhs += row;
            }
            nvtxRangePushA("Linear System");
            /*--------------
            LU Decomposition
            --------------*/
            d_ptr_A = h_ptr_A;
            cublas_check(hipblasZgetrfBatched(cublasHandle[tid], row_sub[i], thrust::raw_pointer_cast(d_ptr_A.data()), row_sub[i], NULL, d_ptr_solverInfo, batchSize));
            /*-----------
            Solve x = A\b
            -----------*/
            d_ptr_rhs = h_ptr_rhs;
            cublas_check(hipblasZgetrsBatched(cublasHandle[tid], HIPBLAS_OP_N, row_sub[i], 1, thrust::raw_pointer_cast(d_ptr_A.data()), row_sub[i], NULL,
                                             thrust::raw_pointer_cast(d_ptr_rhs.data()), row_sub[i], &solverInfo_solve, batchSize));
            /*-----------------
            Synchronize Streams
            -----------------*/
            hipStreamSynchronize(streams[tid]);
        } // matrix loop
    } // omp parallel
    nvtxRangePop();

    timerLoop.stop();
    nvtxRangePop();

    std::cout << ">> Matrix loop finished" << std::endl;
    std::cout << ">>>> Time taken (s) = " << timerLoop.getDurationMicroSec()*1e-6 << "\n" << std::endl;

    // Copy solution from device to host
    thrust::host_vector<hipDoubleComplex> rhs = d_rhs;


    io::writeSolVecComplex(rhs, filepath_sol, filename_sol);
/*
    thrust::host_vector<hipDoubleComplex> A = d_A;
    io::writeSolVecComplex(A, filepath_sol, "A.dat");
*/

    // Destroy cuBLAS & streams
    for (size_t i = 0; i < num_threads; ++i){
        hipblasDestroy(cublasHandle[i]);
        hipStreamDestroy(streams[i]);
    }

    timerTotal.stop();
    std::cout << ">>>>>> Total execution time (s) = " << timerTotal.getDurationMicroSec()*1e-6 << "\n" << std::endl;
}
