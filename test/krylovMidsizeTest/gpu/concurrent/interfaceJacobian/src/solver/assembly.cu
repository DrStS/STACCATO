#include "hip/hip_runtime.h"
// Libraries
#include <iostream>
#include <cassert>

// hipComplex
#include <hip/hip_complex.h>

// cuBLAS
#include <hipblas.h>

// Header Files
#include "assembly.cuh"
#include "../helper/helper.cuh"

// Namespace
using namespace staccato;

__global__ void assembleGlobalMatrix4Batched_kernel(
    hipDoubleComplex * __restrict__ const d_ptr_A,
    const hipDoubleComplex * __restrict__ const d_ptr_K,
    const hipDoubleComplex * __restrict__ const d_ptr_M,
    const int nnz_sub, const double freq_square)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx < nnz_sub )
    {
        const hipDoubleComplex k = d_ptr_K[idx];
        hipDoubleComplex A = d_ptr_M[idx];
        A.x *= freq_square;
        A.y *= freq_square;
        A.x += k.x;
        A.y += k.y;
        d_ptr_A[idx] = A;
    }
}

// Assembles global matrix for batched execution
void assembly::assembleGlobalMatrixBatched(hipStream_t stream, hipDoubleComplex *d_ptr_A,
                                           hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                           int nnz_sub, double freq_square){
    constexpr int block_size = 128;
    assembleGlobalMatrix4Batched_kernel<<<((nnz_sub-1)/block_size)+1,block_size,0,stream>>>(d_ptr_A,d_ptr_K,d_ptr_M,nnz_sub,freq_square);
    hipError_t cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
}
