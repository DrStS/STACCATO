#include "hip/hip_runtime.h"
// Libraries
#include <iostream>
#include <cassert>
#include <stdio.h>
#include <algorithm>

// hipComplex
#include <hip/hip_complex.h>

// cuBLAS
#include <hipblas.h>

// Header Files
#include "assembly.cuh"
#include "../helper/helper.cuh"

__global__ void assembleGlobalMatrixBatched_kernel(hipDoubleComplex ** __restrict__ const d_ptr_A, const hipDoubleComplex * __restrict__ const d_ptr_K,
                                                   const hipDoubleComplex * __restrict__ const d_ptr_M, const int nnz_sub, const int *freq_square,
                                                   const int batchSize, const int num_matrix, const int num_blocks){
    // Get thread index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_thread_local = threadIdx.x;
    int idx_block = blockIdx.x;
    int freq_shift = 0;

    printf("%i", idx_thread_local);

    if (idx_thread_local < nnz_sub){
        for (size_t i = 0; i < (int)batchSize/num_blocks; ++i){
            const hipDoubleComplex k = d_ptr_K[idx_thread_local];
            hipDoubleComplex A = d_ptr_M[idx_thread_local];
            A.x *= freq_square[idx_block + freq_shift];
            A.y *= freq_square[idx_block + freq_shift];
            A.x += k.x;
            A.y += k.y;
            freq_shift += num_blocks;
            d_ptr_A[idx_block + freq_shift][idx_thread_local] = A;

            printf("hello");
        }
    }
}

// Assembles global matrix for batched execution
void assembly::assembleGlobalMatrixBatched(hipStream_t stream, hipDoubleComplex **d_ptr_A,
                                           hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                           int nnz_sub, int *freq_square, const int batchSize, const int num_matrix){
    constexpr int block = 320;                              // Number of threads per block
    //int grid = std::min(32, (nnz_sub*batchSize)/block + 1); // Number of blocks per grid
    int grid = (nnz_sub*batchSize)/block + 1; // Number of blocks per grid
    assembleGlobalMatrixBatched_kernel <<< grid, block >>> (d_ptr_A, d_ptr_K, d_ptr_M, nnz_sub, freq_square, batchSize, num_matrix, grid);
    hipError_t cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
}
