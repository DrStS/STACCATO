#include "hip/hip_runtime.h"
// Libraries
#include <iostream>
#include <vector>
#include <string>
#include <cmath>
#include <algorithm>

// OpenMP
#include <omp.h>

// THRUST
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// CUBLAS
#include <hipblas.h>

// CUSPARSE
#include <hipsparse.h>

// Header files
#include "io/io.cuh"
#include "helper/Timer.cuh"
#include "helper/helper.cuh"
#include "helper/math.cuh"

// Definitions
#define	PI	3.14159265359

int main (int argc, char *argv[]){

	// Command line arguments
	if (argc < 5){
		std::cerr << ">> Usage: " << argv[0] << " -f <maximum frequency> -m <matrix repetition>" << std::endl;
		std::cerr << ">> NOTE: There are 12 matrices and matrix repetition increases the total number of matrices (e.g. matrix repetition of 5 will use 60 matrices)" << std::endl;
		std::cerr << "         Frequency starts from 1 to maximum frequency" << std::endl;
		return 1;
	}

	double freq_max = atof(argv[2]);
	int mat_repetition = atoi(argv[4]);
	int num_matrix = mat_repetition*12;
	std::cout << ">> Maximum Frequency: " << freq_max << std::endl;
	std::cout << ">> Total number of matrices: " << num_matrix << "\n" << std::endl;

	// Vector of filepaths
	std::string filepath[2];
	filepath[0] = "/opt/software/examples/MOR/r_approx_180/\0";
	filepath[1] = "/opt/software/examples/MOR/r_approx_300/\0";

	// Solution filepath
	std::string filepath_sol = "output/";

	// Solution filename
	std::string filename_sol = "solution.dat";

	// Array of matrix sizes (row)
	int row_baseline[] = {126, 132, 168, 174, 180, 186, 192, 288, 294, 300, 306, 312};

	// Array of filenames
	std::string baseName_K = "KSM_Stiffness_r\0";
	std::string baseName_M = "KSM_Mass_r\0";
	std::string baseName_D = "KSM_Damping_r\0";
	std::string base_format = ".mtx\0";
	std::string filename_K[12];
	std::string filename_M[12];
	std::string filename_D[12];

	// Parameters
	bool isComplex = 1;
	double freq, freq_square;
	double freq_min = 1;
	const double alpha = 4*PI*PI;
	hipDoubleComplex one;			// Dummy scailing factor for global matrix assembly
	one.x = 1;
	one.y = 0;
	hipDoubleComplex rhs_val;
	rhs_val.x = (double)1.0;
	rhs_val.y = (double)0.0;

	// OpenMP
	int num_threads = num_matrix;
	omp_set_num_threads(num_threads);

	timerTotal.start();

	// Library initialisation
	hipblasStatus_t hipblasStatus_t;
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	hipsparseStatus_t cusparseStatus;
	hipsparseHandle_t cusparseHandle;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	// Create matrix host_vectors
	thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub(12);
	thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub(12);
	thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub(12);

	// Read and process MTX file
	for (size_t i = 0; i < 7; i++){
		filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
		filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
		filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
		io::readMtxDense(K_sub[i], filepath[0], filename_K[i], isComplex);
		io::readMtxDense(M_sub[i], filepath[0], filename_M[i], isComplex);
		io::readMtxDense(D_sub[i], filepath[0], filename_D[i], isComplex);
		K_sub[i].pop_back();
		M_sub[i].pop_back();
		D_sub[i].pop_back();
	}

	for (size_t i = 7; i < 12; i++){
		filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
		filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
		filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
		io::readMtxDense(K_sub[i], filepath[1], filename_K[i], isComplex);
		io::readMtxDense(M_sub[i], filepath[1], filename_M[i], isComplex);
		io::readMtxDense(D_sub[i], filepath[1], filename_D[i], isComplex);
		K_sub[i].pop_back();
		M_sub[i].pop_back();
		D_sub[i].pop_back();
	}
	std::cout << ">> Matrices imported" << std::endl;

	// Get matrix sizes
	thrust::host_vector<int> row_sub(num_matrix);
	thrust::host_vector<int> size_sub(num_matrix);
	thrust::host_vector<size_t> ptr_mat_shift(num_matrix);
	thrust::host_vector<size_t> ptr_vec_shift(num_matrix);
	int nnz = 0;
	int row = 0;
	size_t idx;
	for (size_t j = 0; j < mat_repetition; j++){
		for (size_t i = 0; i < 12; i++){
			idx = i + 12*j;
			row_sub[idx] = row_baseline[i];
			size_sub[idx] = row_sub[i]*row_sub[i];
			ptr_mat_shift[idx] = nnz;
			ptr_vec_shift[idx] = row;
			nnz += size_sub[idx];
			row  += row_sub[idx];
		}
	}

	// Combine matrices into a single array
	thrust::host_vector<hipDoubleComplex> K(nnz);
	thrust::host_vector<hipDoubleComplex> M(nnz);
	thrust::host_vector<hipDoubleComplex> D(nnz);
	auto K_sub_ptr = &K_sub[0];
	auto M_sub_ptr = &M_sub[0];
	auto D_sub_ptr = &D_sub[0];
	size_t array_shift = 0;
	for (size_t j = 0; j < mat_repetition; j++){
		for (size_t i = 0; i < 12; i++){
			K_sub_ptr = &K_sub[i];
			M_sub_ptr = &M_sub[i];
			D_sub_ptr = &D_sub[i];
			thrust::copy(K_sub_ptr->begin(), K_sub_ptr->end(), K.begin() + array_shift);
			thrust::copy(M_sub_ptr->begin(), M_sub_ptr->end(), M.begin() + array_shift);
			thrust::copy(D_sub_ptr->begin(), D_sub_ptr->end(), D.begin() + array_shift);
			array_shift += size_sub[i];
		}
	}

	std::cout <<">> Matrices combined\n" << std::endl;

	// Generate CSR format
	timerAux.start();
	thrust::host_vector<int> csrRowPtr(row+1);
	thrust::host_vector<int> csrColInd(nnz);
	generateCSR(csrRowPtr, csrColInd, row_sub, size_sub, row, nnz, num_matrix);
	thrust::device_vector<int> d_csrRowPtr = csrRowPtr;
	thrust::device_vector<int> d_csrColInd = csrColInd;
	timerAux.stop();
	std::cout <<">> CSR Format Generated" << std::endl;
	std::cout <<">>>> Time taken = " << timerAux.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

	// Send matrices to device
	timerMatrixCpy.start();
	thrust::device_vector<hipDoubleComplex> d_K = K;
	thrust::device_vector<hipDoubleComplex> d_M = M;
	thrust::device_vector<hipDoubleComplex> d_D = D;

	timerMatrixCpy.stop();
	std::cout << ">> Matrices copied to device " << std::endl;
	std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

	// Create RHS directly on device
	timerMatrixCpy.start();
	thrust::device_vector<hipDoubleComplex> d_rhs(row, rhs_val);
	//thrust::device_vector<hipDoubleComplex> d_rhs_buf = d_rhs;
	timerMatrixCpy.stop();
	std::cout << ">> RHS copied to device " << std::endl;
	std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

	// Create matrix device_vectors
	thrust::device_vector<hipDoubleComplex> d_A(nnz);

	// Get raw pointers to matrices
	hipDoubleComplex *d_ptr_K 	 = thrust::raw_pointer_cast(d_K.data());
	hipDoubleComplex *d_ptr_M 	 = thrust::raw_pointer_cast(d_M.data());
	hipDoubleComplex *d_ptr_D 	 = thrust::raw_pointer_cast(d_D.data());
	hipDoubleComplex *d_ptr_A 	 = thrust::raw_pointer_cast(d_A.data());

	// Get raw pointers to CSR arrays
	int *d_ptr_csrRowPtr = thrust::raw_pointer_cast(d_csrRowPtr.data());
	int *d_ptr_csrColInd = thrust::raw_pointer_cast(d_csrColInd.data());

	// Get raw pointers to RHS vectors
	hipDoubleComplex *d_ptr_rhs = thrust::raw_pointer_cast(d_rhs.data());

	// Create solution vector on host
	thrust::host_vector<hipDoubleComplex> sol(row*freq_max);

	// Create solution vector on device
	thrust::device_vector<hipDoubleComplex> d_z(row);				// Intermediate solution
	thrust::device_vector<hipDoubleComplex> d_sol(row*freq_max);		// Final solution

	// Get raw pointers to solution vector
	hipDoubleComplex *d_ptr_z   = thrust::raw_pointer_cast(d_z.data());
	hipDoubleComplex *d_ptr_sol = thrust::raw_pointer_cast(d_sol.data());

	timerMatrixComp.start();
	// M = 4*pi^2*M (Single computation suffices)
	hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &alpha, d_ptr_M, 1);
	assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);
	timerMatrixComp.stop();
	std::cout << ">> M_tilde computed with cuBLAS" << std::endl;
	std::cout << ">>>> Time taken = " << timerMatrixComp.getDurationMicroSec()*1e-6 << " (sec)\n" << std::endl;

	// Stream initialisation
	const int num_streams = num_threads;
	hipStream_t streams[num_streams];
	for (size_t i = 0; i < num_streams; i++) hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);

	/*-----------------------------
	LU Decomposition initialisation
	-----------------------------*/
	timerAux.start();
	// Matrix Descriptions
	hipsparseMatDescr_t descr_A, descr_L, descr_U;
	hipsparseCreateMatDescr(&descr_A);
	hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseCreateMatDescr(&descr_L);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);
	hipsparseCreateMatDescr(&descr_U);
	hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	// Solver Infos
	csrilu02Info_t solverInfo_A;
	csrsv2Info_t solverInfo_L, solverInfo_U;
	hipsparseCreateCsrilu02Info(&solverInfo_A);
	hipsparseCreateCsrsv2Info(&solverInfo_L);
	hipsparseCreateCsrsv2Info(&solverInfo_U);
	// Transpose operations
	const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const hipsparseOperation_t trans_U = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	// Solver policies
	const hipsparseSolvePolicy_t policy_A = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	// Zero Pivoting
	int structural_zero, numerical_zero;
	// Buffer space
	int bufferSize_A, bufferSize_L, bufferSize_U, bufferSize;
	cusparseStatus = hipsparseZcsrilu02_bufferSize(cusparseHandle, row, nnz, descr_A, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_A, &bufferSize_A);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
	cusparseStatus = hipsparseZcsrsv2_bufferSize(cusparseHandle, trans_L, row, nnz, descr_L, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_L, &bufferSize_L);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
	cusparseStatus = hipsparseZcsrsv2_bufferSize(cusparseHandle, trans_U, row, nnz, descr_U, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_U, &bufferSize_U);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
	bufferSize = std::max(bufferSize_A, std::max(bufferSize_L, bufferSize_U));
	thrust::device_vector<int> d_buffer(bufferSize);
	void* d_ptr_buffer = thrust::raw_pointer_cast(d_buffer.data());
	// Perform analysis
	cusparseStatus = hipsparseZcsrilu02_analysis(cusparseHandle, row, nnz, descr_A, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_A, policy_A, d_ptr_buffer);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
	cusparseStatus = hipsparseXcsrilu02_zeroPivot(cusparseHandle, solverInfo_A, &structural_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
	cusparseStatus = hipsparseZcsrsv2_analysis(cusparseHandle, trans_L, row, nnz, descr_L, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_L, policy_L, d_ptr_buffer);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
	cusparseStatus = hipsparseZcsrsv2_analysis(cusparseHandle, trans_U, row, nnz, descr_U, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_U, policy_U, d_ptr_buffer);
	assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
	timerAux.stop();
	std::cout << ">> LU decomposition initialised" << std::endl;
	std::cout << ">>>> Time taken (s) = " << timerAux.getDurationMicroSec()*1e-6 << "\n" << std::endl;

	/*------------
	Frequency Loop
	------------*/
	timerLoop.start();
	int sol_shift = 0;
	for (size_t it = (size_t)freq_min; it <= (size_t)freq_max; it++){
		// Compute scaling
		freq = (double)it;
		freq_square = -(freq*freq);

		/*-----------------------------------------------
		// Assemble global matrix ( A = K - f^2*M_tilde )
		-----------------------------------------------*/
		d_A = d_M;
		// Scale A with -f^2
		hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &freq_square, d_ptr_A, 1);
		assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);
		// Sum A with K
		hipblasStatus_t = hipblasZaxpy(cublasHandle, nnz, &one, d_ptr_K, 1, d_ptr_A, 1);
		assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

		/*--------------
		LU Decomposition
		--------------*/
		cusparseStatus = hipsparseZcsrilu02(cusparseHandle, row, nnz, descr_A, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_A, policy_A, d_ptr_buffer);
		assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
		cusparseStatus = hipsparseXcsrilu02_zeroPivot(cusparseHandle, solverInfo_A, &numerical_zero);
		if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);

		/*-----------
		Solve x = A\b
		-----------*/
		// Solve z = L\b
		cusparseStatus = hipsparseZcsrsv2_solve(cusparseHandle, trans_L, row, nnz, &one, descr_L, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_L,
												d_ptr_rhs, d_ptr_z, policy_L, d_ptr_buffer);
		assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
		// Solve x = U\z
		cusparseStatus = hipsparseZcsrsv2_solve(cusparseHandle, trans_U, row, nnz, &one, descr_U, d_ptr_A, d_ptr_csrRowPtr, d_ptr_csrColInd, solverInfo_U,
												d_ptr_z, d_ptr_sol+sol_shift, policy_U, d_ptr_buffer);
		assert(HIPSPARSE_STATUS_SUCCESS == cusparseStatus);
		// Update solution vector shift
		sol_shift += row;
	}
	timerLoop.stop();

	std::cout << ">>>> Frequency loop finished" << std::endl;
	std::cout << ">>>>>> Time taken (s) = " << timerLoop.getDurationMicroSec()*1e-6 << "\n" << std::endl;

	sol = d_sol;

	// Write out solution vectors
	io::writeSolVecComplex(sol, filepath_sol, filename_sol);

	// Destroy cuBLAS & cuSparse
	hipblasDestroy(cublasHandle);
	hipsparseDestroy(cusparseHandle);

	// Destroy streams
	for (size_t i = 0; i < num_streams; i++) hipStreamDestroy(streams[i]);

	timerTotal.stop();
	std::cout << ">>>>>> Total execution time (s) = " << timerTotal.getDurationMicroSec()*1e-6 << "\n" << std::endl;
}
