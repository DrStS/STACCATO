#include "hip/hip_runtime.h"
//Libraries
#include <iostream>
#include <iomanip>
#include <hip/hip_complex.h>

//Header Files
#include "config.cuh"

// Namespace
using namespace staccato;

void config::configureTest(int argc, char *argv[], double &freq_max, int &mat_repetition, int &subComponents, int &num_streams, int &num_threads, int &batchSize){
    // Usage
    if (argc < 5){
        std::cerr << ">> Usage: " << argv[0] << " -f <maximum frequency> -m <matrix repetition> -stream <number of CUDA streams> -batch <batch size>" << std::endl;
        std::cerr << ">> NOTE: There are 12 matrices and matrix repetition increases the total number of sub-components (e.g. matrix repetition of 5 will use 60 sub-components)" << std::endl;
        std::cerr << "         Frequency starts from 1 to maximum frequency" << std::endl;
        std::cerr << "         Default number of CUDA streams is 1" << std::endl;
        std::cerr << "         Default number of batch size is freq max (currently only supports batchSize = freq_max)" << std::endl;
        std::exit(1);
    }
    // Set parameters
    freq_max = atof(argv[2]);
    mat_repetition = atoi(argv[4]);
    subComponents = mat_repetition*12;
    num_streams = 1;
    if (argc > 6) num_streams = atoi(argv[6]);
    num_threads = num_streams;
    batchSize = freq_max;
    if (argc > 8) batchSize = atoi(argv[8]);
    // Output messages
    std::cout << ">> Maximum Frequency: " << freq_max << std::endl;
    std::cout << ">> Total number of sub-components: " << subComponents << std::endl;
    std::cout << ">> Number of CUDA streams: " << num_streams << std::endl;
    std::cout << ">> Number of batched matrices: " << batchSize << "\n" << std::endl;
}

void config::check_memory(int mat_repetition, double freq_max, int num_threads){
    /*-----------------
    MEMORY REQUIREMENTS
    -----------------*/
    /*
    1. K, M, D = nnz * 3 * mat_repetition
    2. B, C = nnz_B * 2 * mat_repetition
    3. rhs (sol) = row * freq_max
    4. H = freq_max * nnz_H
    5. A = nt * freq_max * nnz_max
    6. B_batch, C_batch = nt * freq_max * nnz_max_B * 2
    7. d_ptr_K, d_ptr_M, d_ptr_D, d_ptr_B, d_ptr_C = subComponents * 5
    8. d_ptr_A_batch, d_ptr_rhs, d_ptr_B_batch, d_ptr_C_batch, d_ptr_H = freq_max * 5
    */
    unsigned int memory_nnz, memory_nnz_B, memory_row, memory_nnz_H, memory_nnz_max, memory_nnz_max_B, memory_ptr;
    memory_nnz       = sizeof(hipDoubleComplex) * 611424;       // 1
    memory_nnz_B     = sizeof(hipDoubleComplex) * 120060;       // 2
    memory_row       = sizeof(hipDoubleComplex) * 2658;         // 3
    memory_nnz_H     = sizeof(hipDoubleComplex) * 23400;        // 4
    memory_nnz_max   = sizeof(hipDoubleComplex) * 97344;        // 5
    memory_nnz_max_B = sizeof(hipDoubleComplex) * 22464;        // 6
    memory_ptr       = sizeof(hipDoubleComplex*);               // 7
    double memory_required = (
                              memory_nnz*3*mat_repetition +
                              memory_nnz_B*2*mat_repetition +
                              memory_row*freq_max +
                              memory_nnz_H*freq_max +
                              num_threads*freq_max*memory_nnz_max +
                              num_threads*freq_max*memory_nnz_max_B*2 +
                              memory_ptr * 12 * mat_repetition * 5 +
                              memory_ptr * freq_max * 5
                             )*1E-9;

    if (memory_required > 32){
        std::cerr << ">> NOT ENOUGH MEMORY ON GPU" << std::endl;
        std::cerr << ">>>> Memory Required = " << std::setprecision(3) << memory_required << "GB" << std::endl;
        std::cerr << ">>>> Hardware Limit = 32GB" << std::endl;
        std::exit(1);
    }
    else std::cout << ">> Memory Required = " << std::setprecision(3) << memory_required << "GB\n" << std::endl;
}
