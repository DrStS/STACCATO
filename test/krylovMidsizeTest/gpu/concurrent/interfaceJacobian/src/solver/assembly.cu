#include "hip/hip_runtime.h"
// Libraries
#include <iostream>
#include <cassert>

// hipComplex
#include <hip/hip_complex.h>

// cuBLAS
#include <hipblas.h>

// Header Files
#include "assembly.cuh"
#include "../helper/helper.cuh"

// Namespace
using namespace staccato;

__global__ void assembleGlobalMatrixBatched_kernel(hipDoubleComplex ** __restrict__ const d_ptr_A_batch, const hipDoubleComplex * __restrict__ const d_ptr_K,
                                                    const hipDoubleComplex * __restrict__ const d_ptr_M, const int nnz_sub, const int * __restrict__ const freq_square,
                                                    const int batchSize)
{
    // Thread indices
    int idx_thread_global = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_thread_freq   = (int)idx_thread_global/nnz_sub;
    int idx_thread_K_M    = idx_thread_global - idx_thread_freq*nnz_sub;
    // Total size of array batch
    int nnz_batch = nnz_sub * batchSize;
    //extern __shared__ int freq_squared_shared[];

    if (idx_thread_global < nnz_batch){
        //freq_shared[idx_thread_freq] = freq_square[idx_thread_freq];
        const hipDoubleComplex k = d_ptr_K[idx_thread_K_M];
        hipDoubleComplex A = d_ptr_M[idx_thread_K_M];
        A.x *= -freq_square[idx_thread_freq];
        A.y *= -freq_square[idx_thread_freq];
        //A.x *= -freq_squared_shared[idx_thread_freq];
        //A.y *= -freq_squared_shared[idx_thread_freq];
        A.x += k.x;
        A.y += k.y;
        d_ptr_A_batch[idx_thread_freq][idx_thread_K_M] = A;
    }
}

__global__ void constructMatricesBatched_kernel(hipDoubleComplex * __restrict__ d_ptr_B, hipDoubleComplex * __restrict__ d_ptr_C,
                                                hipDoubleComplex ** __restrict__ d_ptr_B_batch, hipDoubleComplex ** __restrict__ d_ptr_C_batch,
                                                const int nnz_sub_B, const int batchSize)
{
    // Thread indices
    int idx_thread_global = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_thread_freq   = (int)idx_thread_global/nnz_sub_B;
    int idx_thread_B_C    = idx_thread_global - idx_thread_freq*nnz_sub_B;
    // Total size of array batch
    int nnz_batch = nnz_sub_B * batchSize;

    if (idx_thread_global < nnz_batch){
        const hipDoubleComplex b = d_ptr_B[idx_thread_B_C];
        const hipDoubleComplex c = d_ptr_C[idx_thread_B_C];
        d_ptr_B_batch[idx_thread_freq][idx_thread_B_C] = b;
        d_ptr_C_batch[idx_thread_freq][idx_thread_B_C] = c;
    }
}

// Assembles global matrix for batched execution
void assembly::assembleGlobalMatrixBatched(hipStream_t stream, hipDoubleComplex **d_ptr_A_batch,
                                           hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                           const int nnz_sub, const int *freq_square, const int batchSize)
{
    constexpr int block = 1024;                         // Number of threads per block
    int grid = (int)(nnz_sub*batchSize/block) + 1;      // Number of blocks per grid (sufficient for a grid to cover nnz_sub*batchSize)
    //size_t shared_memory_size = batchSize*sizeof(int);  // Size of shared memory
    //assembleGlobalMatrixBatched_kernel <<< grid, block, shared_memory_size, stream >>> (d_ptr_A_batch, d_ptr_K, d_ptr_M, nnz_sub, freq_square, batchSize);
    assembleGlobalMatrixBatched_kernel <<< grid, block, 0, stream >>> (d_ptr_A_batch, d_ptr_K, d_ptr_M, nnz_sub, freq_square, batchSize);
    hipError_t cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
}

// Constructs matrices needed for Interface Jacobian in batch
void assembly::constructMatricesBatched(hipStream_t stream, hipDoubleComplex *d_ptr_B, hipDoubleComplex *d_ptr_C,
                                        hipDoubleComplex **d_ptr_B_batch, hipDoubleComplex **d_ptr_C_batch,
                                        const int nnz_sub_B, const int batchSize)
{
    constexpr int block = 1024;
    int grid = (int)(nnz_sub_B*batchSize/block) + 1;
    constructMatricesBatched_kernel <<< grid, block, 0, stream >>> (d_ptr_B, d_ptr_C, d_ptr_B_batch, d_ptr_C_batch, nnz_sub_B, batchSize);
    hipError_t cudaStatus = hipGetLastError();
    assert(cudaStatus == hipSuccess);
}
