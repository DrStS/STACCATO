#include "hip/hip_runtime.h"
// Libraries
#include <fstream>
#include <iostream>
#include <string>
#include <ctime>
#include <limits>
#include <iomanip>

// THRUST
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// Header files
#include "io.cuh"

// Reads dense Mtx file
void io::readMtxDense(thrust::host_vector<hipDoubleComplex> &A, std::string _filepath, std::string _filename, bool _isComplex){
	// Local variables
	size_t rowSize, colSize, entrySize;
	double _real, _imag;
	// Open file
	std::ifstream input;
	input.open(_filepath + _filename);
	//input.precision(std::numeric_limits<float>::digits8);
	// Ignore first line
	while (input.peek() == '%') input.ignore(2048, '\n');
	// Get matrix dimension
	input >> rowSize >> colSize;
	entrySize = rowSize * colSize;

	if (!input){
		std::cout << "File not found." << std::endl;
	}
	else {
		std::cout << ">> Reading matrix from "<< _filepath + _filename << " ... " << std::endl;
		std::cout << ">> Matrix size: " << rowSize << " x " << colSize << std::endl;
		A.resize(entrySize+1);	// Causes segmentation fault without +1
		clock_t io_time;
		io_time = clock();
		// Complex matrix
		if (_isComplex){
			std::cout << ">> Matrix type: COMPLEX" << std::endl;
			//thrust::host_vector<hipComplex> A(entrySize);
			//hipComplex *A = (hipComplex*)malloc(entrySize*sizeof(hipComplex));
			int i = 0;
			while (!input.eof()) {
				input >> _real >> _imag;
				hipDoubleComplex temp;
				temp.x = _real;
				temp.y = _imag;
				A[i] = temp;
				i++;
			}
			io_time = clock() - io_time;
			std::cout << ">> Matrix " << _filename << " read" << std::endl;
			std::cout << ">>>> Time taken = " << ((float)io_time)/CLOCKS_PER_SEC << " (sec)" << "\n" << std::endl;
		}
		// Real matrix
		else if (!_isComplex){
			std::cout << ">> Matrix type: REAL" << std::endl;
			//thrust::host_vector<hipComplex> A(entrySize);
			//hipComplex *A = (hipComplex*)malloc(entrySize*sizeof(hipComplex));
			int i = 0;
			while (!input.eof()) {
				input >> _real >> _imag;
				hipDoubleComplex temp;
				temp.x = _real;
				temp.y = _imag;
				A[i] = temp;
				i++;
			}
			io_time = clock() - io_time;
			std::cout << ">> Matrix " << _filename << " read" << std::endl;
			std::cout << ">>>> Time taken = " << ((float)io_time)/CLOCKS_PER_SEC << " (sec)" << "\n" << std::endl;
		}
	}
	input.close();
}

// Writes solution vector
void io::writeSolVecComplex(thrust::host_vector<hipDoubleComplex> &sol, std::string _filepath, std::string _filename){
	std::ofstream output;
	output.open(_filepath + _filename);
	clock_t io_time;
	io_time = clock();
	// Write header
	if (!output.is_open()){
		std::cout << ">> ERROR: Unable to open output file for solution vector" << std::endl;
	}
	else{
		output << std::setw(25) << std::left << "Real" << "    ";
		output << std::setw(25) << std::left << "Imag" << "\r\n";
		// Write data
		for (size_t i = 0; i < sol.size(); i++){
			output << std::setprecision(16) << std::setw(25) << std::left << sol[i].x << "    ";
			output << std::setprecision(16) << std::setw(25) << std::left << sol[i].y << "\r\n";
		}
	}
	// Close file
	output.close();
	io_time = clock() - io_time;
	// Output messages
	std::cout << ">> Solution vector written in " << _filepath + _filename << std::endl;
	std::cout << ">>>> Vector size = " << sol.size() << std::endl;
	std::cout << ">>>> Time taken = " << ((float)io_time)/CLOCKS_PER_SEC << " (sec)" << "\n" << std::endl;
}
