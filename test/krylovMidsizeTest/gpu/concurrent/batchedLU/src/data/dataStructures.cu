// Libraries
#include <string>
#include <thrust/host_vector.h>

// Header Files
#include "dataStructures.cuh"
#include "../io/io.cuh"

// Namespace
using namespace staccato;

void data::getInfoHostDataStructure(
                                    thrust::host_vector<int> &shift_local_A, thrust::host_vector<int> &shift_local_rhs,
                                    thrust::host_vector<int> &row_sub, thrust::host_vector<int> &nnz_sub, int &nnz, int &row, int &nnz_max, int mat_repetition, int row_baseline[]
                                   )
{
    // Get matrix sizes and local shifts
    nnz = 0;
    row = 0;
    size_t idx;
    int mat_shift = 0;
    int sol_shift = 0;
    for (size_t j = 0; j < mat_repetition; ++j){
        for (size_t i = 0; i < 12; ++i){
            // Index for combined matrix
            idx = i + 12*j;
            // Sub-component matrix & vector sizes
            row_sub[idx] = row_baseline[i];
            nnz_sub[idx] = row_sub[i]*row_sub[i];
            // Accumulate total matrix & vector sizes
            nnz += nnz_sub[idx];
            row += row_sub[idx];
            // (Local) shifts for each sub-components from combined matrix
            shift_local_A[idx]   = mat_shift;
            shift_local_rhs[idx] = sol_shift;
            // Update shifts
            mat_shift += nnz_sub[idx];
            sol_shift += row_sub[idx];
        }
    }
    // Get maximum matrix size
    auto nnz_max_it = thrust::max_element(nnz_sub.begin(), nnz_sub.end());
    nnz_max = *nnz_max_it;
}

void data::combineHostMatrices(
                               thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub,
                               thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub,
                               thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub,
                               thrust::host_vector<hipDoubleComplex> &K, thrust::host_vector<hipDoubleComplex> &M, thrust::host_vector<hipDoubleComplex> &D,
                               int nnz, int mat_repetition, thrust::host_vector<int> nnz_sub
                              )
{
    K.resize(nnz);
    M.resize(nnz);
    D.resize(nnz);
    // Combine matrices into a single array
    auto K_sub_ptr = &K_sub[0];
    auto M_sub_ptr = &M_sub[0];
    auto D_sub_ptr = &D_sub[0];
    size_t array_shift = 0;
    for (size_t j = 0; j < mat_repetition; ++j){
        for (size_t i = 0; i < 12; ++i){
            K_sub_ptr = &K_sub[i];
            M_sub_ptr = &M_sub[i];
            D_sub_ptr = &D_sub[i];
            thrust::copy(K_sub_ptr->begin(), K_sub_ptr->end(), K.begin() + array_shift);
            thrust::copy(M_sub_ptr->begin(), M_sub_ptr->end(), M.begin() + array_shift);
            thrust::copy(D_sub_ptr->begin(), D_sub_ptr->end(), D.begin() + array_shift);
            array_shift += nnz_sub[i];
        }
    }
    std::cout <<">> Matrices combined" << std::endl;
}

void data::constructHostDataStructure(
                                      std::string filename_K[], std::string filename_M[], std::string filename_D[], std::string filepath[],
                                      std::string baseName_K,   std::string baseName_M,   std::string baseName_D,   std::string base_format,
                                      int row_baseline[],
                                      thrust::host_vector<thrust::host_vector<hipDoubleComplex>> &K_sub,
                                      thrust::host_vector<thrust::host_vector<hipDoubleComplex>> &M_sub,
                                      thrust::host_vector<thrust::host_vector<hipDoubleComplex>> &D_sub,
                                      thrust::host_vector<int> &shift_local_A, thrust::host_vector<int> &shift_local_rhs,
                                      thrust::host_vector<int> &row_sub, thrust::host_vector<int> &nnz_sub,
                                      int &nnz, int &row, int &nnz_max, int mat_repetition,
                                      thrust::host_vector<hipDoubleComplex> &K, thrust::host_vector<hipDoubleComplex> &M, thrust::host_vector<hipDoubleComplex> &D
                                     )
{
    /*------------------------
    READ AND PROCESS MTX FILES
    ------------------------*/
    for (size_t i = 0; i < 7; ++i){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[0], filename_K[i], true);
        io::readMtxDense(M_sub[i], filepath[0], filename_M[i], true);
        io::readMtxDense(D_sub[i], filepath[0], filename_D[i], true);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }
    for (size_t i = 7; i < 12; ++i){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[1], filename_K[i], true);
        io::readMtxDense(M_sub[i], filepath[1], filename_M[i], true);
        io::readMtxDense(D_sub[i], filepath[1], filename_D[i], true);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }
    std::cout << ">> Matrices imported" << std::endl;

    /*-------------
    GET MATRIX INFO
    -------------*/
    data::getInfoHostDataStructure(shift_local_A, shift_local_rhs, row_sub, nnz_sub, nnz, row, nnz_max, mat_repetition, row_baseline);

    /*----------------------------------
    COMBINE MATRICES INTO A SINGLE ARRAY
    ----------------------------------*/
    data::combineHostMatrices(K_sub, M_sub, D_sub, K, M, D, nnz, mat_repetition, nnz_sub);
}

