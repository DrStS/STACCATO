// Libraries
#include <iostream>
#include <cassert>

// hipComplex
#include <hip/hip_complex.h>

// cuBLAS
#include <hipblas.h>

// Header Files
#include "assembly.cuh"
#include "../helper/helper.cuh"

// Assembles global matrix ( A = K - f^2*M_tilde )
void assembly::assembleGlobalMatrix(hipStream_t stream, hipblasStatus_t hipblasStatus_t, hipblasHandle_t cublasHandle,
                                    hipDoubleComplex *d_ptr_A, hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                    int nnz, int mat_shift, hipDoubleComplex one, double freq_square){
    // Copy M to A
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZcopy(cublasHandle, nnz, d_ptr_M, 1, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Scale M by f^2
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &freq_square, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Sum A with K
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZaxpy(cublasHandle, nnz, &one, d_ptr_K, 1, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
}

// Assembles global matrix for batched execution
void assembly::assembleGlobalMatrix4Batched(hipblasHandle_t cublasHandle, hipDoubleComplex *d_ptr_A,
                                            hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                            int nnz_sub, hipDoubleComplex one, double freq_square){
    // Copy M to A
    cublas_check(hipblasZcopy(cublasHandle, nnz_sub, d_ptr_M, 1, d_ptr_A, 1));
    // Scale M by f^2
    cublas_check(hipblasZdscal(cublasHandle, nnz_sub, &freq_square, d_ptr_A, 1));
    // Sum A with K
    cublas_check(hipblasZaxpy(cublasHandle, nnz_sub, &one, d_ptr_K, 1, d_ptr_A, 1));
}
