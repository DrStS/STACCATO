// Libraries
#include <iostream>
#include <cassert>

// hipComplex
#include <hip/hip_complex.h>

// cuBLAS
#include <hipblas.h>

// Header Files
#include "assembly.cuh"

// Assembles global matrix ( A = K - f^2*M_tilde )
void assembly::assembleGlobalMatrix(hipStream_t stream, hipblasStatus_t hipblasStatus_t, hipblasHandle_t cublasHandle,
                                    hipDoubleComplex *d_ptr_A, hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                    int nnz, int mat_shift, hipDoubleComplex one, double freq_square){
    // Copy M to A
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZcopy(cublasHandle, nnz, d_ptr_M, 1, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Scale M by f^2
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &freq_square, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Sum A with K
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZaxpy(cublasHandle, nnz, &one, d_ptr_K, 1, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
}

// Assembles global matrix for batched execution
void assembly::assembleGlobalMatrix4Batched(hipStream_t stream, hipblasStatus_t hipblasStatus_t, hipblasHandle_t cublasHandle,
                                            hipDoubleComplex *d_ptr_A, hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                            int nnz_sub, hipDoubleComplex one, double freq_square){
    // Copy M to A
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZcopy(cublasHandle, nnz_sub, d_ptr_M, 1, d_ptr_A, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Scale M by f^2
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZdscal(cublasHandle, nnz_sub, &freq_square, d_ptr_A, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Sum A with K
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZaxpy(cublasHandle, nnz_sub, &one, d_ptr_K, 1, d_ptr_A, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
}
