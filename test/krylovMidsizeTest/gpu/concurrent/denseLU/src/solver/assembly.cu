// Libraries
#include <iostream>
#include <cassert>

// hipComplex
#include <hip/hip_complex.h>

// cuBLAS
#include <hipblas.h>

// Header Files
#include "assembly.cuh"

// Assembles global matrix ( A = K - f^2*M_tilde )
void assembly::assembleGlobalMatrix(int tid, hipStream_t stream, hipblasStatus_t hipblasStatus_t, hipblasHandle_t cublasHandle,
                                    hipDoubleComplex *d_ptr_A, hipDoubleComplex *d_ptr_K, hipDoubleComplex *d_ptr_M,
                                    int nnz, int mat_shift, hipDoubleComplex one, double freq_square){
    // Copy M to A
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZcopy(cublasHandle, nnz, d_ptr_M, 1, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Scale M by f^2
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZdscal(cublasHandle, nnz, &freq_square, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
    // Sum A with K
    hipblasSetStream(cublasHandle, stream);
    hipblasStatus_t = hipblasZaxpy(cublasHandle, nnz, &one, d_ptr_K, 1, d_ptr_A + mat_shift, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);
}
