// Libraries
#include <iostream>
#include <string>
#include <cmath>

// OpenMP
#include <omp.h>

// THRUST
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

// CUCOMPLEX
#include <hip/hip_complex.h>

// CUBLAS
#include <hipblas.h>

// NVTX: https://devblogs.nvidia.com/cuda-pro-tip-generate-custom-application-profile-timelines-nvtx/
#include <nvToolsExt.h>

// Header files
#include "config/config.cuh"
#include "io/io.cuh"
#include "solver/assembly.cuh"
#include "helper/Timer.cuh"
#include "helper/helper.cuh"

// Definitions
#define	PI	3.14159265359
#define MAX_NUM_THREADS 32

// Pinned Allocators
typedef thrust::system::cuda::experimental::pinned_allocator<int> pinnedAllocInt;
typedef thrust::system::cuda::experimental::pinned_allocator<hipDoubleComplex> pinnedAlloc;
typedef thrust::system::cuda::experimental::pinned_allocator<hipDoubleComplex*> pinnedAllocPtr;

int main (int argc, char *argv[]){

    /*--------------------
    COMMAND LINE ARGUMENTS
    --------------------*/
    double freq_max;
    int mat_repetition, subComponents, num_streams, num_threads, batchSize;
    // Configure test environment with command line arguments
    config::configureTest(argc, argv, freq_max, mat_repetition, subComponents, num_streams, num_threads, batchSize);

    /*---------------------
    FILEPATHS AND FILENAMES
    ---------------------*/
    // Vector of filepaths
    std::string filepath[2];
    filepath[0] = "/opt/software/examples/MOR/r_approx_180/\0";
    filepath[1] = "/opt/software/examples/MOR/r_approx_300/\0";
    // Solution filepath
    std::string filepath_sol = "output/";
    // Solution filename
    std::string filename_sol = "solution.dat";
    // Array of filenames
    std::string baseName_K = "KSM_Stiffness_r\0";
    std::string baseName_M = "KSM_Mass_r\0";
    std::string baseName_D = "KSM_Damping_r\0";
    std::string base_format = ".mtx\0";
    std::string filename_K[12];
    std::string filename_M[12];
    std::string filename_D[12];
    std::string filename_input[12];

    /*--------
    PARAMETERS
    --------*/
    bool isComplex = 1;
    double freq, freq_square;
    const double alpha = 4*PI*PI;
    hipDoubleComplex rhs_val;
    rhs_val.x = (double)1.0;
    rhs_val.y = (double)0.0;
    // Array of matrix sizes (row)
    int row_baseline[] = {126, 132, 168, 174, 180, 186, 192, 288, 294, 300, 306, 312};
    // Sort the array row_baseline
    //std::sort(row_baseline.begin(), row_baseline.end());

    /*----------------------------
    OPENMP & CUBLAS INITIALIZATION
    ----------------------------*/
    // OpenMP
    int tid;
    omp_set_num_threads(num_threads);
    // cuBLAS
    timerTotal.start();
    hipblasHandle_t cublasHandle[MAX_NUM_THREADS];
    for (size_t i = 0; i < num_threads; ++i) hipblasCreate(cublasHandle + i);

    /*-----------------------
    CHECK MEMORY REQUIREMENTS
    -----------------------*/
    config::check_memory(mat_repetition, freq_max, num_threads);

    /*-------------
    DATA STRUCTURES
    --------------*/
    // Create matrix host_vectors
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> K_sub(12);
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> M_sub(12);
    thrust::host_vector<thrust::host_vector<hipDoubleComplex>> D_sub(12);

    // Read and process MTX file
    for (size_t i = 0; i < 7; ++i){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[0], filename_K[i], isComplex);
        io::readMtxDense(M_sub[i], filepath[0], filename_M[i], isComplex);
        io::readMtxDense(D_sub[i], filepath[0], filename_D[i], isComplex);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }

    for (size_t i = 7; i < 12; ++i){
        filename_K[i] = baseName_K + std::to_string(row_baseline[i]) + base_format;
        filename_M[i] = baseName_M + std::to_string(row_baseline[i]) + base_format;
        filename_D[i] = baseName_D + std::to_string(row_baseline[i]) + base_format;
        io::readMtxDense(K_sub[i], filepath[1], filename_K[i], isComplex);
        io::readMtxDense(M_sub[i], filepath[1], filename_M[i], isComplex);
        io::readMtxDense(D_sub[i], filepath[1], filename_D[i], isComplex);
        K_sub[i].pop_back();
        M_sub[i].pop_back();
        D_sub[i].pop_back();
    }
    std::cout << ">> Matrices imported" << std::endl;

    // Get matrix sizes
    thrust::host_vector<int> row_sub(subComponents);
    thrust::host_vector<int> nnz_sub(subComponents);
    int nnz = 0;
    int row = 0;
    size_t idx;
    for (size_t j = 0; j < mat_repetition; ++j){
        for (size_t i = 0; i < 12; ++i){
            idx = i + 12*j;
            row_sub[idx] = row_baseline[i];
            nnz_sub[idx] = row_sub[i]*row_sub[i];
            nnz += nnz_sub[idx];
            row += row_sub[idx];
        }
    }

    // Get maximum matrix size
    auto nnz_max_it = thrust::max_element(nnz_sub.begin(), nnz_sub.end());
    int nnz_max = *nnz_max_it;

    // Combine matrices into a single array on host (to make use of GPU's high bandwidth. We could also import the matrices directly like this)
    thrust::host_vector<hipDoubleComplex> K(nnz);
    thrust::host_vector<hipDoubleComplex> M(nnz);
    thrust::host_vector<hipDoubleComplex> D(nnz);
    auto K_sub_ptr = &K_sub[0];
    auto M_sub_ptr = &M_sub[0];
    auto D_sub_ptr = &D_sub[0];
    size_t array_shift = 0;
    for (size_t j = 0; j < mat_repetition; ++j){
        for (size_t i = 0; i < 12; ++i){
            K_sub_ptr = &K_sub[i];
            M_sub_ptr = &M_sub[i];
            D_sub_ptr = &D_sub[i];
            thrust::copy(K_sub_ptr->begin(), K_sub_ptr->end(), K.begin() + array_shift);
            thrust::copy(M_sub_ptr->begin(), M_sub_ptr->end(), M.begin() + array_shift);
            thrust::copy(D_sub_ptr->begin(), D_sub_ptr->end(), D.begin() + array_shift);
            array_shift += nnz_sub[i];
        }
    }

    std::cout <<">> Matrices combined" << std::endl;

    /*----------------------
    DATA STRUCTURES (DEVICE)
    ----------------------*/
    nvtxRangePushA("Data Structures (Device)");
    // Send matrices to device
    timerMatrixCpy.start();
    thrust::device_vector<hipDoubleComplex> d_K = K;
    thrust::device_vector<hipDoubleComplex> d_M = M;
    thrust::device_vector<hipDoubleComplex> d_D = D;
    timerMatrixCpy.stop();
    std::cout << ">> Matrices copied to device " << std::endl;
    std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

    // Create RHS directly on device
    timerMatrixCpy.start();
    thrust::device_vector<hipDoubleComplex> d_rhs(row*freq_max, rhs_val);
    timerMatrixCpy.stop();
    std::cout << ">> RHS copied to device " << std::endl;
    std::cout << ">>>> Time taken = " << timerMatrixCpy.getDurationMicroSec()*1e-6 << " (sec)" << "\n" << std::endl;

    // Create matrix device_vectors
    thrust::device_vector<hipDoubleComplex> d_A(num_threads*freq_max*nnz_max);

    // Get vector of raw pointers to matrices
    hipDoubleComplex *d_ptr_K_base = thrust::raw_pointer_cast(d_K.data());
    hipDoubleComplex *d_ptr_M_base = thrust::raw_pointer_cast(d_M.data());
    hipDoubleComplex *d_ptr_D_base = thrust::raw_pointer_cast(d_D.data());

    // Get local shifts
    thrust::host_vector<hipDoubleComplex*> h_ptr_K(subComponents);
    thrust::host_vector<hipDoubleComplex*> h_ptr_M(subComponents);
    thrust::host_vector<hipDoubleComplex*> h_ptr_D(subComponents);
    size_t mat_shift = 0;
    size_t sol_shift = 0;
    thrust::host_vector<int> shift_local_rhs(subComponents);
    thrust::host_vector<int> shift_local_A(subComponents);
    for (size_t i = 0; i < subComponents; ++i){
        h_ptr_K[i] = d_ptr_K_base + mat_shift;
        h_ptr_M[i] = d_ptr_M_base + mat_shift;
        h_ptr_D[i] = d_ptr_D_base + mat_shift;
        shift_local_rhs[i] = sol_shift;
        shift_local_A[i]   = mat_shift;
        mat_shift += nnz_sub[i];
        sol_shift += row_sub[i];
    }

    // Get raw pointers to matrix A and rhs
    hipDoubleComplex *d_ptr_A_base = thrust::raw_pointer_cast(d_A.data());
    hipDoubleComplex *d_ptr_rhs_base = thrust::raw_pointer_cast(d_rhs.data());
    nvtxRangePop();

    timerMatrixComp.start();
    // M = 4*pi^2*M (Single computation suffices)
    cublas_check(hipblasZdscal(cublasHandle[0], nnz, &alpha, d_ptr_M_base, 1));
    timerMatrixComp.stop();
    std::cout << ">> M_tilde computed with cuBLAS" << std::endl;
    std::cout << ">>>> Time taken = " << timerMatrixComp.getDurationMicroSec()*1e-6 << " (sec)\n" << std::endl;

    // Solver Info for batched LU decomposition
    thrust::device_vector<int> d_solverInfo(batchSize);
    int *d_ptr_solverInfo = thrust::raw_pointer_cast(d_solverInfo.data());
    int solverInfo_solve;

    // Stream initialisation
    hipStream_t streams[num_streams];
    for (size_t i = 0; i < num_streams; ++i){
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
        std::cout << ">> Stream " << i << " created" << std::endl;
    }

    /*--------------------
    Krylov Subspace Method
    --------------------*/
    nvtxRangePushA("Krylov Subspace Method");
    timerLoop.start();
    std::cout << "\n>> Matrix loop started for batched execution" << std::endl;
#pragma omp parallel private(tid, freq, freq_square) num_threads(num_threads)
    {
        // Get thread number
        tid = omp_get_thread_num();
        // Allocate vector of array pointers to A in each thread
        thrust::device_vector<hipDoubleComplex*> d_ptr_A(batchSize);
        thrust::device_vector<hipDoubleComplex*> d_ptr_rhs(batchSize);
        thrust::host_vector<hipDoubleComplex*, pinnedAllocPtr> h_ptr_A(batchSize);
        thrust::host_vector<hipDoubleComplex*, pinnedAllocPtr> h_ptr_rhs(batchSize);
        // Initialise shifts
        int shift_global_A, shift_batch_A, shift_global_rhs;
        shift_global_A = tid*freq_max*nnz_max;
        // Set cuBLAS stream
        hipblasSetStream(cublasHandle[tid], streams[tid]);
    // Loop over each matrix size
    #pragma omp for
        for (size_t i = 0; i < subComponents; ++i){
            /*---------------------------------------------------------------
            Assemble Global Matrix & Update pointers to each matrix A and RHS
            ---------------------------------------------------------------*/
            // Initialise Shifts
            shift_global_rhs = 0;
            shift_batch_A = 0;
            // Loop over batch (assume batchSize = freq_max)
            for (size_t j = 0; j < batchSize; ++j){
                // Update matrix A pointer
                h_ptr_A[j] = d_ptr_A_base + shift_batch_A + shift_global_A;
                // Update rhs pointer
                h_ptr_rhs[j] = d_ptr_rhs_base + shift_local_rhs[i] + shift_global_rhs;
                // Compute frequency (assume batchSize = freq_max)
                freq = (j+1);
                freq_square = -(freq*freq);
                // Assemble matrix
                nvtxRangePushA("Matrix Assembly");
                assembly::assembleGlobalMatrixBatched(streams[tid], h_ptr_A[j], h_ptr_K[i], h_ptr_M[i], nnz_sub[i], freq_square);
                nvtxRangePop();
                // Update shifts
                shift_batch_A    += nnz_sub[i];
                shift_global_rhs += row;
            }
            nvtxRangePushA("Linear System");
            /*--------------
            LU Decomposition
            --------------*/
            d_ptr_A = h_ptr_A;
            cublas_check(hipblasZgetrfBatched(cublasHandle[tid], row_sub[i], thrust::raw_pointer_cast(d_ptr_A.data()), row_sub[i], NULL, d_ptr_solverInfo, batchSize));
            /*-----------
            Solve x = A\b
            -----------*/
            d_ptr_rhs = h_ptr_rhs;
            cublas_check(hipblasZgetrsBatched(cublasHandle[tid], HIPBLAS_OP_N, row_sub[i], 1, thrust::raw_pointer_cast(d_ptr_A.data()), row_sub[i], NULL,
                                             thrust::raw_pointer_cast(d_ptr_rhs.data()), row_sub[i], &solverInfo_solve, batchSize));
            /*-----------------
            Synchronize Streams
            -----------------*/
            hipStreamSynchronize(streams[tid]);
        } // matrix loop
    } // omp parallel
    nvtxRangePop();

    timerLoop.stop();
    nvtxRangePop();

    std::cout << ">> Matrix loop finished" << std::endl;
    std::cout << ">>>> Time taken (s) = " << timerLoop.getDurationMicroSec()*1e-6 << "\n" << std::endl;

    // Copy solution from device to host
    thrust::host_vector<hipDoubleComplex> rhs = d_rhs;

/*
    io::writeSolVecComplex(rhs, filepath_sol, filename_sol);
    thrust::host_vector<hipDoubleComplex> A = d_A;
    io::writeSolVecComplex(A, filepath_sol, "A.dat");
*/

    // Destroy cuBLAS & streams
    for (size_t i = 0; i < num_threads; ++i){
        hipblasDestroy(cublasHandle[i]);
        hipStreamDestroy(streams[i]);
    }

    timerTotal.stop();
    std::cout << ">>>>>> Total execution time (s) = " << timerTotal.getDurationMicroSec()*1e-6 << "\n" << std::endl;
}
